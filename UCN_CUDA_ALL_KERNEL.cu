#include "hip/hip_runtime.h"

/* Include files ----------------------------------------------------------------- */
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// #include "cuPrintf.cuh"
// #include "cuPrintf.cu"

/* 

    ###########################################################
    # This is version 2.0
    # CUDA Runge-Kutta 4th/5th Order
    #
    # Author : Spencer McBride Day Moore
    # Date   : May 15, 2015
    ###########################################################

    This is a standalone nvcc compiled program that implements a function in order
    to be implemented with double precision floating point numbers compute capability
    of CUDA device must be 2.0 or higher (-arch=sm_20 is the flag that goes with nvcc 
    for 2.0, -arch=sm_21 for 2.1 and so on). the #define variables are listed below 
    the header files.  Also the function fxn_dydx is the example given is only allowed
    single variable dependence in addition to time f_x(t,x)=dx/dt, f_y(t,y)=dy/dt etc. 
    and not f_x(t,x,y,z)=dx/dt etc..  This will need to be changed for usefulness in 
    the neutron transport code.  Currently the user can pass variables to the 
    individual threads via the constant double arrays cu_A and cu_B.  They are currently
    initialized ot random values between positive and negative one for testing purposes. 
    There is no adaptive step implemented although the stepwise error is outputted along
    with (t,x,y,z).
*/

#include "UCN_CUDA_ALL_KERNEL.cuh"

__constant__ int d_CONST_INT[e_d_CONST_INT_LAST];
__constant__ double d_CONST[e_d_CONST_LAST];

__device__ int CUDA_ipow( int base, int exp)
{
	int result = -1;
	if (exp<0) result = 0;
	else if (exp==0) result = 1;
	else if (exp>1)
	{
		result = 1;
		for (int i = exp; i>0; i--)
		{
				result *= base;
		}
	}
  return result;
}
__device__ int getGlobal_blockId_3D()
{
	int blockId = blockIdx.x + 
		(blockIdx.y * gridDim.x)	+ (
		gridDim.x * gridDim.y * blockIdx.z);
	return blockId;
}
__device__ int getGlobalIdx_3D_3D()
{
	int blockId = getGlobal_blockId_3D();
	int threadId = 
		(blockId * blockDim.x * blockDim.y * blockDim.z) + 
		threadIdx.z * (blockDim.x * blockDim.y) + 
		threadIdx.y * blockDim.x + 
		threadIdx.x;
	return threadId;
	// int vi_RETURN_VALUE = ( blockIdx.x * blockDim.x + threadIdx.x );
	// return vi_RETURN_VALUE
}
__device__ int fvi_ISOLATE_INT_RANGE( int vi_INPUT, int vi_MSB, int vi_LSB)
{
	int vi_ABS_INPUT = vi_INPUT < 0 ? -vi_INPUT : vi_INPUT;
	int vi_INDEX, vi_MOD;
	for (vi_INDEX = 1, vi_MOD = 1 ; vi_INDEX<vi_MSB; vi_INDEX++)
	{
		vi_MOD = vi_MOD * 2;
	}
	int vi_INTERMEDIATE = ( vi_ABS_INPUT % vi_MOD );
	int vi_FINAL = ( vi_INTERMEDIATE >> ( vi_LSB - 1 ) );
	if ( vi_FINAL != ( vi_INTERMEDIATE * CUDA_ipow(2, vi_LSB - 1))) vi_FINAL = -1;
	return vi_FINAL;
}
__device__ /*__host__*/ double CUDA_theta( double x, double y, double z)
{
  double temp;
  if ( (z!=0) && ((x*x+y*y)!=0) ) temp = atan2(sqrt(x*x+y*y),z);
  else if (z==0) temp = d_CONST[e_d_CONST_def_PI]/2.0;
  else temp = d_CONST[e_d_CONST_def_TINY];	
  if (temp >= d_CONST[e_d_CONST_def_PI]) temp=2*d_CONST[e_d_CONST_def_PI]-temp;	
  return(temp);
}
__device__ /*__host__*/ double CUDA_ro( double x, double y)
{
	double solution;
  if (x*x+y*y != 0) solution = sqrt(x*x + y*y);
  else solution = ( d_CONST[e_d_CONST_def_TINY] );
	return solution;
}
__device__ /*__host__*/ double CUDA_r( double x, double y, double z)
{
  double solution;
  if ((x*x+y*y+z*z)!= 0) solution = (sqrt(x*x+y*y+z*z));
  else solution = ( d_CONST[e_d_CONST_def_TINY] );
  return solution;
}
__device__ /*__host__*/ double CUDA_phi( double x, double y)
{
  double solution; 
  if (x!=0)
    solution = atan2(y,x);
  else
    solution = ((d_CONST[e_d_CONST_def_PI]/2.0));
  return solution;
}
__device__ /*__host__*/ double CUDA_SPIN_X( double spinor_0, double spinor_1, double spinor_2, double spinor_3 )
{
	double xhat1;
  if (d_CONST[e_d_CONST_INT_FLAG_CLASSICAL_SPIN]==0) xhat1 = spinor_1;
	else if (d_CONST[e_d_CONST_INT_FLAG_CLASSICAL_SPIN]==1)
	{
		double snorm = (spinor_0*spinor_0) + 	(spinor_1*spinor_1) + (spinor_2*spinor_2) + (spinor_3*spinor_3);
		xhat1 =( 2.*((spinor_0*spinor_2) + (spinor_1*spinor_3)))/snorm;
	}
	return xhat1;
}
__device__ /*__host__*/ double CUDA_SPIN_Y( double spinor_0, double spinor_1, double spinor_2, double spinor_3 )
{
	double yhat1;
  if (d_CONST[e_d_CONST_INT_FLAG_CLASSICAL_SPIN]==0) yhat1 = spinor_2;
	else if (d_CONST[e_d_CONST_INT_FLAG_CLASSICAL_SPIN]==1)\
	{
		double snorm = (spinor_0*spinor_0) + (spinor_1*spinor_1) +  
						(spinor_2*spinor_2) + (spinor_3*spinor_3);
		yhat1 = ( 2.*((spinor_0*spinor_3) - (spinor_1*spinor_2)))/snorm;
	}
  return yhat1;
}
__device__ /*__host__*/ double CUDA_SPIN_Z( double spinor_0, double spinor_1, double spinor_2, double spinor_3 )
{
  double zhat1;
  if (d_CONST[e_d_CONST_INT_FLAG_CLASSICAL_SPIN]==0) zhat1 = spinor_3;
	else if (d_CONST[e_d_CONST_INT_FLAG_CLASSICAL_SPIN]==1)
	{
		double snorm = (spinor_0*spinor_0) + (spinor_1*spinor_1) +  
          (spinor_2*spinor_2) + (spinor_3*spinor_3);
		zhat1 = ((spinor_0*spinor_0) + (spinor_1*spinor_1) - 
	   (spinor_2*spinor_2) - (spinor_3*spinor_3))/snorm;
	}
	return zhat1;
}
__device__ /*__host__*/ int CUDA_SPIN_XVS_XYZ( 
	double spin[],
	double xvs[])
{	
	spin[0] = CUDA_SPIN_X(xvs[6], xvs[7], xvs[8], xvs[9]);
	spin[1] = CUDA_SPIN_Y(xvs[6], xvs[7], xvs[8], xvs[9]);
	spin[2] = CUDA_SPIN_Z(xvs[6], xvs[7], xvs[8], xvs[9]);
}
__device__ /*__host__*/ double CUDA_SPIN_XVS_SINGLE( 
	double xvs[],
	int vi_X0_Y1_Z2)
{
	double vd_RETURN_VALUE;
	if (vi_X0_Y1_Z2==0) vd_RETURN_VALUE = CUDA_SPIN_X(xvs[6], xvs[7], xvs[8], xvs[9]);
	if (vi_X0_Y1_Z2==1) vd_RETURN_VALUE = CUDA_SPIN_Y(xvs[6], xvs[7], xvs[8], xvs[9]);
	if (vi_X0_Y1_Z2==2) vd_RETURN_VALUE = CUDA_SPIN_Z(xvs[6], xvs[7], xvs[8], xvs[9]);
	return vd_RETURN_VALUE;
}
__device__ /*__host__*/ double CUDA_polcalc_XVS( 
	double bb1x, 
	double bb1y, 
	double bb1z, 
	double xvs[])
{
  double xhat1,yhat1,zhat1,snorm;
  /* const double xhat2,yhat2,zhat2; */
  double bxhat,byhat,bzhat;
  double bnorm,polarization;
  /* const double spinnorm,rnorm,rtest; */

  bnorm = sqrt(bb1x*bb1x + bb1y*bb1y + bb1z*bb1z);
  bxhat = bb1x/bnorm;
  byhat = bb1y/bnorm;
  bzhat = bb1z/bnorm;
	
  xhat1 = 0.;
  yhat1 = 0.;
  zhat1 = 0.;
	// CUDA_SPIN_XVS_XYZ( spin, xvs );
  snorm = (xvs[6]*xvs[6]) + (xvs[7]*xvs[7]) +  
          (xvs[8]*xvs[8]) + (xvs[9]*xvs[9]);
  xhat1 =( 2.*((xvs[6]*xvs[8]) + (xvs[7]*xvs[9])))/snorm;
  yhat1 = ( 2.*((xvs[6]*xvs[9]) - (xvs[7]*xvs[8])))/snorm;
  zhat1 = ((xvs[6]*xvs[6]) + (xvs[7]*xvs[7]) - 
	   (xvs[8]*xvs[8]) - (xvs[9]*xvs[9]))/snorm;
  polarization = xhat1*bxhat + yhat1*byhat + zhat1*bzhat;
  return(polarization);
}
__device__ /*__host__*/ int CUDA_Mag_ROHM_HOLLEY_XVS( double xvs[], double BField[] )
{
	double xi, yi, zi, l, zoff, zoff2, zcent, a0, rcoil, lcoil, rsol, bmax, b0, bpr, bpp, bppp, zrf, lsol,sinth0,costh0, bc0, sinthc0, costhc0, bcc0;
	zoff = 0.295;
  zoff2 = 1.595;
  rsol = .08;
  b0   = .975372;/* .93562; */

  a0 = .21;

  zcent = 0.1641;
  bmax = 7.0327;
  rcoil = .144;
  lcoil = .097;

  bpr  = 0.10749; 
  bpp = -0.102;  /*T/m^2*/
  bppp = 3.45;  /* T/m^3 */

  zrf = 1.14;
  lsol = (zoff2-zoff)/2.;
  sinth0 = rsol/sqrt(rsol*rsol+lsol*lsol);
  costh0 = lsol*sinth0/rsol;
  bc0 = b0/costh0;
 
  sinthc0 = rcoil/sqrt(rcoil*rcoil + lcoil*lcoil);
  costhc0 = lcoil*sinthc0/rcoil;
  bcc0  = bmax/costhc0;
  double rho;


  double zd1, anm,anmsq,rhnm1;
  double costh1,sinth1,costh2,sinth2;
  double sinth1sq,sinth2sq;
  double rhnm2;
  int result;

  double sinthc1,costhc1,sinthc2,costhc2;
  double sinthc1sq,sinthc2sq;
  double rhnmc;

  /*
  double sinthn1,costhn1,sinthn2,costhn2;
  double sinthn1sq,sinthn2sq;
  double rhnmn;
  */

  /* compute common expressions 
   (this is for legibility as well as optimization) */
  sinth1 = rsol/sqrt(rsol*rsol+(zoff2-xvs[2])*(zoff2-xvs[2]));
  costh1 = (zoff2 - xvs[2])*sinth1/rsol;

  sinth2 = rsol/sqrt(rsol*rsol+(xvs[2]-zoff)*(xvs[2]-zoff));
  costh2 = (xvs[2] - zoff)*sinth2/rsol;

  sinth1sq = sinth1*sinth1;
  sinth2sq = sinth2*sinth2;

  rho = sqrt(xvs[0]*xvs[0]+xvs[1]*xvs[1]);
  rhnm2 = rho/rsol;
  rhnmc = rho/rcoil;
  /*  rhnmn = rho/rnew; */

  sinthc1 = rcoil/sqrt(rcoil*rcoil+(zcent+lcoil-xvs[2])*(zcent+lcoil-xvs[2]));
  costhc1 = (zcent+lcoil - xvs[2])*sinthc1/rcoil;

  sinthc2 = rcoil/sqrt(rcoil*rcoil + (xvs[2] - (zcent - lcoil))*(xvs[2] - (zcent - lcoil)));
  costhc2 = (xvs[2]-(zcent - lcoil))*sinthc2/rcoil;

  sinthc1sq = sinthc1*sinthc1;
  sinthc2sq = sinthc2*sinthc2;
  /*
  sinthn1 = rnew/sqrt(rnew*rnew+(zcnew+lnew-xvs[2])*(zcnew+lnew-xvs[2]));
  costhn1 = (zcnew+lnew - xvs[2])*sinthn1/rnew;

  sinthn2 = rnew/sqrt(rnew*rnew + (xvs[2] - (zcnew - lnew))*(xvs[2] - (zcnew - lnew)));
  costhn2 = (xvs[2]-(zcnew - lnew))*sinthn2/rnew;

  sinthn1sq = sinthn1*sinthn1;
  sinthn2sq = sinthn2*sinthn2;
  */
  zd1 = sqrt(a0*a0+(xvs[2]-zcent)*(xvs[2]-zcent));
  anm = a0/zd1;
  anmsq = anm*anm;
  rhnm1 = rho/zd1;
  result = 0;
  BField[0] = bcc0*xvs[0]*(sinthc1*sinthc1sq-sinthc2*sinthc2sq)/(4.*rcoil)
    /*    + bcn0*xvs[0]*(sinthn1*sinthn1sq-sinthn2*sinthn2sq)/(4.*rnew)  */
    + bc0*xvs[0]*(sinth1*sinth1sq-sinth2*sinth2sq)/(4.*rsol)
    - xvs[0]*bpr/2. - xvs[0]*(xvs[2]-zrf)*bpp/2. + xvs[0]*(rho*rho/16. -(xvs[2]-zrf)*(xvs[2]-zrf)/4.)*bppp;
  BField[1] =  bcc0*xvs[1]*(sinthc1*sinthc1sq-sinthc2*sinthc2sq)/(4.*rcoil)
    /*   + bcn0*xvs[1]*(sinthn1*sinthn1sq-sinthn2*sinthn2sq)/(4.*rnew) */
    + bc0*xvs[1]*(sinth1*sinth1sq-sinth2*sinth2sq)/(4.*rsol)
    - xvs[1]*bpr/2. - xvs[1]*(xvs[2]-zrf)*bpp/2. + xvs[1]*(rho*rho/16. -(xvs[2]-zrf)*(xvs[2]-zrf)/4.)*bppp;
  BField[2] =  bcc0*((costhc1+costhc2)/2.+
		     .375*rhnmc*rhnmc*(sinthc1sq*sinthc1sq*costhc1+sinthc2sq*sinthc2sq*costhc2))
    /*    + bcn0*((costhn1+costhn2)/2.+
	  .375*rhnmn*rhnmn*(sinthn1sq*sinthn1sq*costhn1+sinthn2sq*sinthn2sq*costhn2)) */
    + bc0*((costh1+costh2)/2.
	   +.375*rhnm2*rhnm2*(sinth1sq*sinth1sq*costh1+sinth2sq*sinth2sq*costh2))
    + bpr*(xvs[2]-zrf) + ((xvs[2]-zrf)*(xvs[2]-zrf)/2. - rho*rho/4.)*bpp + ((xvs[2]-zrf)*((xvs[2]-zrf)*(xvs[2]-zrf)/6.-rho*rho/4.))*bppp;
 
 return( result );
}


__device__ /*__host__*/ int CUDA_dB_ROHM_HOLLEY_XVS( double xvs[], double BField[], double dField_1D_FLAT[])
{
	double xi, yi, zi, l, zoff, zoff2, zcent, a0, rcoil, lcoil, rsol, bmax, b0, bpr, bpp, bppp, zrf, lsol,sinth0,costh0, bc0, sinthc0, costhc0, bcc0;
	zoff = 0.295;
  zoff2 = 1.595;
  rsol = .08;
  b0   = .975372;/* .93562; */

  a0 = .21;

  zcent = 0.1641;
  bmax = 7.0327;
  rcoil = .144;
  lcoil = .097;

  bpr  = 0.10749; 
  bpp = -0.102;  /*T/m^2*/
  bppp = 3.45;  /* T/m^3 */

  zrf = 1.14;
  lsol = (zoff2-zoff)/2.;
  sinth0 = rsol/sqrt(rsol*rsol+lsol*lsol);
  costh0 = lsol*sinth0/rsol;
  bc0 = b0/costh0;
 
  sinthc0 = rcoil/sqrt(rcoil*rcoil + lcoil*lcoil);
  costhc0 = lcoil*sinthc0/rcoil;
  bcc0  = bmax/costhc0;	
  double rho;   
  double zd1sq,anmsq,rhnm1;
  double costh1,sinth1,costh2,sinth2;
  double sinth1sq,sinth2sq;
  double  rhnm2;

  double costhc1,sinthc1,costhc2,sinthc2;
  double sinthc1sq,sinthc2sq;
  double  rhnmc2;
  double BZ;
  double Bslope, Bdelta, Bsigma;
  
  int result;


  /*  !!!!!!! NOT UP TO DATE   !!!!!   */


  rho = sqrt(xvs[0]*xvs[0]+xvs[1]*xvs[1]);

  sinth1 = rsol/sqrt(rsol*rsol+(zoff2-xvs[2])*(zoff2-xvs[2]));
  costh1 = (zoff2 - xvs[2])*sinth1/rsol;
  sinth2 = rsol/sqrt(rsol*rsol+(xvs[2]-(zoff2-2.*lsol))*(xvs[2]-(zoff2-2.*lsol)));
  costh2 = (xvs[2]-(zoff2-2.*lsol))*sinth2/rsol;
  sinth1sq = sinth1*sinth1;
  sinth2sq = sinth2*sinth2;
 
  rhnm2 = rho/rsol;

  sinthc1 = rcoil/sqrt(rcoil*rcoil+(zcent+lcoil-xvs[2])*(zcent+lcoil-xvs[2]));
  costhc1 = (zcent + lcoil - xvs[2])*sinthc1/rcoil;
  sinthc2 = rcoil/sqrt(rcoil*rcoil+(xvs[2]-(zcent-lcoil))*(xvs[2]-(zcent-lcoil)));
  costhc2 = (xvs[2]-(zcent-lcoil))*sinthc2/rcoil;
  sinthc1sq = sinthc1*sinthc1;
  sinthc2sq = sinthc2*sinthc2;
  
  rhnmc2 = rho/rcoil;  

  zd1sq = (a0*a0+(xvs[2]-zcent)*(xvs[2]-zcent));   /* ! */
  rhnm1 = rho/sqrt(zd1sq);          /* ! */
  anmsq = a0*a0/zd1sq;             /* ! */
    /*	anmsq = anm*anm;*/
  
  BZ = bmax*anmsq*sqrt(anmsq);        /* ! */
  result = 0; /* if direct1,2 <0 or >3, we are at sea. */  
  /*   */
  Bslope = (xvs[2]-zcent)*BZ/zd1sq;      /* ! */
  Bdelta =  bc0*(sinth1*sinth1sq-sinth2*sinth2sq)/(4.*rsol) +
          bcc0*(sinthc1*sinthc1sq-sinthc2*sinthc2sq)/(4.*rcoil);
  /*  fixed sinth4 bug here */
  Bsigma = bc0*(sinth1sq*sinth1sq*costh1+sinth2sq*sinth2sq*costh2) +
    bcc0*(sinthc1sq*sinthc1sq*costhc1+sinthc2sq*sinthc2sq*costhc2);

	/* */
  dField_1D_FLAT[0] = Bdelta - bpr/2. - (xvs[2]-zrf)*bpp/2.+(3.*xvs[0]*xvs[0]+xvs[1]*xvs[1])*bppp/16.;
  dField_1D_FLAT[1] = xvs[0]*xvs[1]*bppp/8.;
  dField_1D_FLAT[2] =  .75 * xvs[0] * Bsigma - xvs[0]*(bpp+(xvs[2]-zrf)*bppp)/2.;
  dField_1D_FLAT[3] =  xvs[0]*xvs[1]*bppp/8.;
  dField_1D_FLAT[4] =  Bdelta - bpr/2. - (xvs[2]-zrf)*bpp/2.+(xvs[0]*xvs[0]+3.*xvs[1]*xvs[1])*bppp/16.;
  dField_1D_FLAT[5] =  .75 * xvs[1] * Bsigma - xvs[1]*(bpp+(xvs[2]-zrf)*bppp)/2.;
  dField_1D_FLAT[6] = - xvs[0]*(bpp+(xvs[2]-zrf)*bppp)/2.;
  dField_1D_FLAT[7] = - xvs[1]*(bpp+(xvs[2]-zrf)*bppp)/2.;
  dField_1D_FLAT[8] =  .5*(bc0/rsol)*( -sinth1*sinth1sq + sinth2*sinth2sq)+ 0.375*(bc0/rsol)
    *(rhnm2*rhnm2)*(sinth1sq*sinth1*(4.*sinth1sq-5.*sinth1sq*sinth1sq))
    - 0.375*(bc0/rsol)*(rhnm2*rhnm2)*(sinth2sq*sinth2*(4.*sinth2sq-5.*sinth2sq*sinth2sq))
    + .5*(bcc0/rcoil)*( -sinthc1*sinthc1sq + sinthc2*sinthc2sq)+ 0.375*(bcc0/rcoil)
    *(rhnmc2*rhnmc2)*(sinthc1sq*sinthc1*(4.*sinthc1sq-5.*sinthc1sq*sinthc1sq))
    - 0.375*(bcc0/rcoil)*(rhnmc2*rhnmc2)*(sinthc2sq*sinthc2*(4.*sinthc2sq-5.*sinthc2sq*sinthc2sq))
    + bpr + (xvs[2]-zrf)*bpp + (2.*(xvs[2]-zrf)*(xvs[2]-zrf)- (xvs[0]*xvs[0]+xvs[1]*xvs[1]))*bppp/4.;
  /* result = dField[comp][part]; */
	/* return(result); */
	return 0;
}
__device__ /*__host__*/ int CUDA_Mag_CONSTANT_XVS( double xvs[], double BField[])
{
	BField[0] = 0.0;
	BField[1] = 0.0;
	BField[2] = 1.0;
	return 0;
}
__device__ /*__host__*/ int CUDA_dB_CONSTANT_XVS( double xvs[], double BField[], double dField_1D_FLAT[])
{
	dField_1D_FLAT[0] = 0.0;
	dField_1D_FLAT[1] = 0.0;
	dField_1D_FLAT[2] = 0.0;
	dField_1D_FLAT[3] = 0.0;
	dField_1D_FLAT[4] = 0.0;
	dField_1D_FLAT[5] = 0.0;
	dField_1D_FLAT[6] = 0.0;
	dField_1D_FLAT[7] = 0.0;
	dField_1D_FLAT[8] = 0.0;
	return 0;
}
__device__ /*__host__*/ int CUDA_Mag_XVS( double xvs[], double BField[])
{
	if (d_CONST_INT[e_d_CONST_INT_FLAG_MAGNETIC]==1) return CUDA_Mag_ROHM_HOLLEY_XVS( xvs, BField );
	else if (d_CONST_INT[e_d_CONST_INT_FLAG_MAGNETIC]==2) return CUDA_Mag_CONSTANT_XVS( xvs, BField );
	else return -1;
}
__device__ /*__host__*/ int CUDA_dB_XVS( double xvs[], double BField[], double dField_1D_FLAT[])
{
	if (d_CONST_INT[e_d_CONST_INT_FLAG_MAGNETIC]==1) return CUDA_dB_ROHM_HOLLEY_XVS( xvs, BField, dField_1D_FLAT);
	else if (d_CONST_INT[e_d_CONST_INT_FLAG_MAGNETIC]==2) return CUDA_dB_CONSTANT_XVS( xvs, BField, dField_1D_FLAT);
	else return -1;
}
__device__ /*__host__*/ int CUDA_RF_BFIELD( double t, double xvs[],	double extra_brf[])
{
		double rfstr = d_CONST[e_d_CONST_RF_BFIELD_MAG];
		double  omega,btoomega,zrf,zdist,zdelta;
		btoomega = d_CONST[e_d_CONST_def_MOMENT]/d_CONST[e_d_CONST_def_HBAR];  // coeff. for psi-dot 
		omega    = 2.0*d_CONST[e_d_CONST_def_MOMENT]*(1.0)/d_CONST[e_d_CONST_def_HBAR]; // reson. frequ. for 1.0 T field 
		zrf      = 1.14; // chosen to match crossing poconst int 
		// some comment explaining this. v 
		zdist = xvs[8];
		zdelta = (zdist - zrf)*(zdist -zrf)/(.05*.05);
		extra_brf[0] = rfstr*cos(omega*(t-0.0)) * (1.0/((1.0+zdelta)*(1.0+zdelta))); 
		extra_brf[1] = 0.;
			// exp(-(zdist-zrf)*(zdist-zrf)/(.05*.05)); 
		extra_brf[2] = 0.;
		
		return 0;
}
__device__ /*__host__*/ int CUDA_derivs_XVS( 
	double t,
	double xvs[],
	double dxvsdt[],
	double BField[],
	double dField_1D_FLAT[])
{	
	double x,y,z;	/* Cartesian coordinates */
  double rad, th;	/* Polar coordinates */
  double B; 	/* Magnitude of B */
  int Bgood;
  double spin[3];
	CUDA_SPIN_XVS_XYZ(spin, xvs);
	x=xvs[0];
  y=xvs[1];
  z=xvs[2];
  rad=CUDA_r(x,y,z);
  th=CUDA_theta(x,y,z);
  dxvsdt[0]=xvs[3]; /* derivative of x = v in x direction */
  dxvsdt[1]=xvs[4]; /* ditto for y */
  dxvsdt[2]=xvs[5]; /* ditto for z */
  dxvsdt[3]=0; /* initializing acceleration in x direction */
  dxvsdt[4]=0; /* ditto for y */
  dxvsdt[5]=0; /* ditto for z */
  Bgood = CUDA_Mag_XVS(xvs, BField);
  B = sqrt( BField[0]*BField[0]+BField[1]*BField[1]+BField[2]*BField[2]);
  Bgood = CUDA_dB_XVS(xvs, BField, dField_1D_FLAT);
	// if (d_CONST_INT[e_d_CONST_INT_def_PERFECT_POLARIZATION] && 
		// isfinite(BField[0]) && 
		// isfinite(BField[1]) && 
		// isfinite(BField[2]) && 
		// (B>0 && isfinite(B)))
	// {
		// spin[0] = BField[0]/B;
		// spin[1] = BField[1]/B;
		// spin[2] = BField[2]/B;
	// }
  if (d_CONST_INT[e_d_CONST_INT_FLAG_MAGNETIC]==1 && B>0 && isfinite(B))
	{
		dxvsdt[3] += - d_CONST[e_d_CONST_def_CORRECTIVE_FACTOR_SPIN]*(d_CONST[e_d_CONST_def_MOMENT_DIV_MASS])*spin[0] / B * 
			(BField[0] * dField_1D_FLAT[0] +     BField[1] * dField_1D_FLAT[1] + BField[2] * dField_1D_FLAT[2]);
		dxvsdt[4] += - d_CONST[e_d_CONST_def_CORRECTIVE_FACTOR_SPIN]*(d_CONST[e_d_CONST_def_MOMENT_DIV_MASS])*spin[1] / B * 
			(BField[0] * dField_1D_FLAT[3] + BField[1] * dField_1D_FLAT[4] +  BField[2] * dField_1D_FLAT[5]);
		dxvsdt[5] += - d_CONST[e_d_CONST_def_CORRECTIVE_FACTOR_SPIN]*(d_CONST[e_d_CONST_def_MOMENT_DIV_MASS])*spin[2] / B * 
			(BField[0] * dField_1D_FLAT[6] + BField[1] * dField_1D_FLAT[7] + BField[2] * dField_1D_FLAT[8]);
  }
	// dxvsdt[3] += - (d_CONST[e_d_CONST_def_MOMENT_DIV_MASS])*spin[0] * 
		// (cos(d_CONST[e_d_CONST_def_HALBACH_K]*xvs[0]) * dField_1D_FLAT[0] + sin(d_CONST[e_d_CONST_def_HALBACH_K]*xvs[0]) * dField_1D_FLAT[1]);
	// dxvsdt[4] += - (d_CONST[e_d_CONST_def_MOMENT_DIV_MASS])*spin[1] * 
		// (cos(d_CONST[e_d_CONST_def_HALBACH_K]*xvs[0]) * dField_1D_FLAT[3] + sin(d_CONST[e_d_CONST_def_HALBACH_K]*xvs[0]) * dField_1D_FLAT[4] );
	// dxvsdt[5] += - (d_CONST[e_d_CONST_def_MOMENT_DIV_MASS])*spin[2] * 
		// (cos(d_CONST[e_d_CONST_def_HALBACH_K]*xvs[0]) * dField_1D_FLAT[6] + sin(d_CONST[e_d_CONST_def_HALBACH_K]*xvs[0]) * dField_1D_FLAT[7] );
	// if (B>0 && isfinite(B)) dxvsdt[4] += exp(-2*d_CONST[e_d_CONST_def_HALBACH_K]*xvs[1])*spin[1]*d_CONST[e_d_CONST_def_HALBACH_K]*d_CONST[e_d_CONST_def_HALBACH_MAX_TESLA]*d_CONST[e_d_CONST_def_HALBACH_MAX_TESLA]*d_CONST[e_d_CONST_def_CORRECTIVE_FACTOR_SPIN]*(d_CONST[e_d_CONST_def_MOMENT_DIV_MASS]);
	/* Gravity is in +/- y direction depending on sign */ 
  if (d_CONST_INT[e_d_CONST_INT_FLAG_GRAVITY]==1) dxvsdt[4]  += d_CONST[e_d_CONST_def_GRAVITY];
	/* Ground spring begins at y=0 and below */
	if (d_CONST_INT[e_d_CONST_INT_FLAG_SPRING]==1)
	{
		if (xvs[1]<0) dxvsdt[4] += -xvs[1]*d_CONST[e_d_CONST_def_SPRING];
	}
	double rfstr = d_CONST[e_d_CONST_RF_BFIELD_MAG];
	double  brf[4],omega,btoomega,zrf,zdist,zdelta;
	btoomega = d_CONST[e_d_CONST_def_MOMENT]/d_CONST[e_d_CONST_def_HBAR];  // coeff. for psi-dot 
	omega    = 2.0*d_CONST[e_d_CONST_def_MOMENT]*(1.0)/d_CONST[e_d_CONST_def_HBAR]; // reson. frequ. for 1.0 T field 
	zrf      = 1.14; // chosen to match crossing poconst int 
	// some comment explaining this. v 
	zdist = spin[2];
	zdelta = (zdist - zrf)*(zdist -zrf)/(.05*.05);
	brf[0] = rfstr*cos(omega*(t-0.0)) * (1.0/((1.0+zdelta)*(1.0+zdelta))); 
	brf[1] = 0.;
		// exp(-(zdist-zrf)*(zdist-zrf)/(.05*.05)); 
	brf[2] = 0.;
	int i;
	double extra_brf[3];
	CUDA_RF_BFIELD(t, xvs, extra_brf);
	if (d_CONST_INT[e_d_CONST_INT_FLAG_RF]==1)
	{
		for ( i = 0; i<3; i++) BField[i] += extra_brf[i];
	}
	
  // Classical Description of Spin 
  // NB moment has "I=1/2" already... 
	dxvsdt[6] = 0.0;
	dxvsdt[7] = btoomega*(xvs[8]*BField[2] - xvs[9]*BField[1]);
	dxvsdt[8] = btoomega*(xvs[9]*BField[0] - xvs[7]*BField[2]);
	dxvsdt[9] = btoomega*(xvs[7]*BField[1] - xvs[8]*BField[0]);
	// ihd/dt = (moment)*(sigma)*B 
  // NB moment has "I=1/2" already... 
  // dxvsdt[6] = btoomega * ( (BField[2] * xvs[7]) + (BField[0] * xvs[9]) - (BField[1] * xvs[8]));
  // dxvsdt[7] = btoomega * (-(BField[2] * xvs[6]) - (BField[0] * xvs[8]) - (BField[1] * xvs[9]));
  // dxvsdt[8] = btoomega * (-(BField[2] * xvs[9]) + (BField[0] * xvs[7]) + (BField[1] * xvs[6]));
  // dxvsdt[9] = btoomega * ( (BField[2] * xvs[8]) - (BField[0] * xvs[6]) + (BField[1] * xvs[7]));
  return 0;
}
__device__ /*__host__*/ int CUDA_rkck_XVS( 
	double *d_IO, 
	int *d_IO_INT, 
	int vi_RECORD,
	double t,
	double xvs[],
	double dxvsdt[],
	double h,
	double xvserr[],
	double xvsout[],
	double BField[], 
	double dField_1D_FLAT[])
{
	// double BField[3], dField_1D_FLAT[9];
  CUDA_derivs_XVS(t, xvs, dxvsdt, BField, dField_1D_FLAT);
  int i;
  /* 	static const a2=(0.2, a3=(0.3, a4=(0.6, a5=(1.0, a6=(0.875; */
  const double b21=(0.2), b31=(3.0/40.0), b32=(9.0/40.0), b41=(0.3), b42=( -0.9), b43=(1.2);
  const double b51=( -11.0/54.0), b52=(2.5), b53=( -70.0/27.0), b54=(35.0/27.0); 
  const double b61=(1631.0/55296.0), b62=(175.0/512.0), b63=(575.0/13824.0), b64=(44275.0/110592.0), b65=(253.0/4096.0); 
  const double c1=(37.0/378.0), c3=(250.0/621.0), c4=(125.0/594.0), c6=(512.0/1771.0);
  const double dc5=( -277.0/14336.0), dc1=(c1-2825.0/27648.0),  dc3=(c3-18575.0/48384.0), dc4=(c4-13525.0/55296.0), dc6=(c6-0.25);
  double ak2[10], ak3[10], ak4[10], ak5[10], ak6[10], xvstemp[10];
  for (i = 0; i<10; i++)	 /* First step */ xvstemp[i]=xvs[i]+b21*h*dxvsdt[i];
  CUDA_derivs_XVS(t, xvs, ak2, BField, dField_1D_FLAT);		/* Second step */
  for (i = 0; i<10; i++) xvstemp[i]=xvs[i]+h*(b31*dxvsdt[i]+b32*ak2[i]);
  CUDA_derivs_XVS(t, xvs, ak3, BField, dField_1D_FLAT);		/* Third step */
  for (i = 0; i<10; i++) xvstemp[i]=xvs[i]+h*(b41*dxvsdt[i]+b42*ak2[i]+b43*ak3[i]);
  CUDA_derivs_XVS(t, xvs, ak4, BField, dField_1D_FLAT);		/* Fourth step */
  for (i = 0; i<10; i++) xvstemp[i]=xvs[i]+h*(b51*dxvsdt[i]+b52*ak2[i]+b53*ak3[i]+b54*ak4[i]);
  CUDA_derivs_XVS(t, xvstemp, ak5, BField, dField_1D_FLAT);		/* Fifth step */
  for (i = 0; i<10; i++) xvstemp[i]=xvs[i]+h*(b61*dxvsdt[i]+b62*ak2[i]+b63*ak3[i]+b64*ak4[i]+b65*ak5[i]);
  CUDA_derivs_XVS(t, xvs, ak6, BField, dField_1D_FLAT);		/* Sixth step */
  for (i = 0; i<10; i++)	xvsout[i]=xvs[i]+h*(c1*dxvsdt[i]+c3*ak3[i]+c4*ak4[i]+c6*ak6[i]); /* Accumulate increments with proper weights */
  for (i = 0; i<10; i++) xvserr[i]=h*(dc1*dxvsdt[i]+dc3*ak3[i]+dc4*ak4[i]+dc5*ak5[i]+dc6*ak6[i]);
  return 0;
}
__device__ /*__host__*/ int CUDA_rkqs_SINGLE_ATTEMPT_XVS(
	double *d_IO, 
	int *d_IO_INT,
	int vi_RECORD, 
	double xvs[],
 	double dxvsdt[],
 	double *t,
 	double htry,
	double *hdid,
 	double *hnext,
 	double xvs_scal[],
 	double BField[],
	double dField_1D_FLAT[],
 	double epsilon[],
 	int *rkqs_TRIED)
{
	int i, j, k;
	double hnext_xv;
	double hnext_s;
  double hcurrent = htry;
	double xvs_temp[10];
  double epsilon_temp[10];
  double error, error_xv, error_s;
  int return_value = -1;
  int return_value_xv = -1;
  int return_value_s = -1;
	double abs_max_xv, abs_max_s;
	double hnext_s_GB, hnext_s_GU, hnext_s_SB, hnext_s_SU;
	double hnext_xv_GB, hnext_xv_GU, hnext_xv_SB, hnext_xv_SU;
	CUDA_rkck_XVS( 
		d_IO, 
		d_IO_INT, 
		vi_RECORD, 
		*t, 
		xvs, 
		dxvsdt, 
		hcurrent, 
		epsilon_temp, 
		xvs_temp, 
		BField, 
		dField_1D_FLAT);
	double epsilon_max_xv = 0.0;
	double epsilon_max_s = 0.0;
	for (i = 0; i<10; i++)
	{
		if (i<6)
		{
			abs_max_xv = fabs(epsilon_temp[i]/xvs_scal[i]);
			// if (abs_max_xv<0) abs_max_xv *= -1.0; 
			if (abs_max_xv>epsilon_max_xv) epsilon_max_xv = abs_max_xv;
			
		}
		else
		{
			abs_max_s = fabs(epsilon_temp[i]/xvs_scal[i]);
			// if (abs_max_s<0) abs_max_s *= -1.0; 
			if (abs_max_s>epsilon_max_s) epsilon_max_s = abs_max_s;
		}
	}
	error_xv = epsilon_max_xv/d_CONST[e_d_CONST_def_MAXERR];
	error_s = epsilon_max_s/d_CONST[e_d_CONST_def_MAXERR1];
	
	/////////////////////////////////////////////////////////////////////////
	/////////////////// XV AND SPIN STEP COMPUTATION ////////////////////////
	/////////////////////////////////////////////////////////////////////////
	
	/* XV: Step succeeded. Compute size of next step */
	hnext_xv_GU = d_CONST[e_d_CONST_def_SAFETY]*hcurrent*pow(error_xv,d_CONST[e_d_CONST_def_PGROW]);
	/* XV: No more than a factor of 5 increase */
	hnext_xv_GB = 5.0*hcurrent; // ((hnext_xv_GU >= 0.0) ? fmin(hnext_xv_GU, 5.0*hcurrent) : fmax(hnext_xv_GU, 5.0*hcurrent));
	/* XV: Truncation error too large. Reduce stepsize */
	hnext_xv_SU = d_CONST[e_d_CONST_def_SAFETY]*hcurrent*pow(error_xv, d_CONST[e_d_CONST_def_PSHRNK]);
	/* XV: No more than a factor of 10 */
	hnext_xv_SB = 0.1*hcurrent; // ((hnext_xv_SU >= 0.0) ? fmax(hnext_xv_SU, 0.1*hcurrent) : fmin(hnext_xv_SU, 0.1*hcurrent));
	/* S: Step succeeded. Compute size of next step */
	hnext_s_GU = d_CONST[e_d_CONST_def_SAFETY1]*hcurrent*pow(error_s,d_CONST[e_d_CONST_def_PGROW1]);
	/* S: No more than a factor of 5 increase */
	hnext_s_GB = 5.0*hcurrent; // ((hnext_s_GU >= 0.0) ? fmin(hnext_s_GU, 5.0*hcurrent) : fmax(hnext_s_GU, 5.0*hcurrent));
	/* S: Truncation error too large. Reduce stepsize */
	hnext_s_SU = d_CONST[e_d_CONST_def_SAFETY1]*hcurrent*pow(error_s, d_CONST[e_d_CONST_def_PSHRNK1]);
	/* S: No more than a factor of 10 */
	hnext_s_SB = 0.1*hcurrent; // ((hcurrent >= 0.0) ? fmax(hnext_s_SU, 0.1*hcurrent) : fmin(hnext_s_SU, 0.1*hcurrent));
	
	/////////////////////////////////////////////////////////////////////////
	/////////////////// XV ERROR AND STEP DECISION //////////////////////////
	/////////////////////////////////////////////////////////////////////////
	
	if (error_xv <= 1.0)
	{
		hnext_xv = ((hcurrent >= 0.0) ? fmin(hnext_xv_GU, hnext_xv_GB) : fmax(hnext_xv_GU, hnext_xv_GB));
    return_value_xv = 0;
  }
	else if (error_xv > 1.0)
	{
		hnext_xv = ((hcurrent >= 0.0) ? fmax(hnext_xv_SU, hnext_xv_SB) : fmin(hnext_xv_SU, hnext_xv_SB));
		return_value_xv = 1;
	}
	
  /////////////////////////////////////////////////////////////////////////
	////////////////// SPIN ERROR AND STEP DECISION /////////////////////////
	/////////////////////////////////////////////////////////////////////////
	
	if (error_s <= 1.0)
	{
		hnext_s = ((hcurrent >= 0.0) ? fmin(hnext_s_GU, hnext_s_GB) : fmax(hnext_s_GU, hnext_s_GB));
		return_value_s = 0;
  }
	else if (error_s > 1.0)
	{
		hnext_s = ((hcurrent >= 0.0) ? fmax(hnext_s_SU, hnext_s_SB) : fmin(hnext_s_SU, hnext_s_SB));
		return_value_s = 1;
	}
	
  /////////////////////////////////////////////////////////////////////////
	/////////////////// FINAL TIME STEP SIGN CHECK //////////////////////////
	/////////////////////////////////////////////////////////////////////////
	
	
	
  /////////////////////////////////////////////////////////////////////////
	/////////////////// FINAL TIME STEP DECISION ////////////////////////////
	/////////////////////////////////////////////////////////////////////////
	
	/* nrerror("stepsize underflow in rkqs"); */ /* diag */
	if ((*t + hcurrent) == *t) return_value = e_RKQS_ERROR_STEPSIZE_UNDERFLOW;
	else if (return_value_xv==0 && return_value_s==0)
	{
		for (i=0; i<10; i++)
		{
			xvs[i] = xvs_temp[i];
			epsilon[i] = epsilon_temp[i];
		}
		*t += hcurrent;
		*hdid = hcurrent;
		*rkqs_TRIED = 0;
		if (hcurrent<=0.0)
		{
			*hnext = fmax ( fmax (hnext_xv, hnext_s) , fmax ( d_CONST[e_d_CONST_tframe], d_CONST[e_d_CONST_tframe_SPIN]));
		}
		else if (hcurrent>0.0) 
		{
			*hnext = fmin ( fmin (hnext_xv, hnext_s) , fmin ( d_CONST[e_d_CONST_tframe], d_CONST[e_d_CONST_tframe_SPIN]));
		}
		return_value = e_RKQS_ERROR_NONE;
	}
	else
	{
		*hdid = 0.0;
		(*rkqs_TRIED)++;
		if (return_value_xv==1 && return_value_s==0) 
		{
			*hnext = hnext_xv;
			return_value = e_RKQS_ERROR_XV_BOUNDS;
		}
		else if (return_value_xv==0 && return_value_s==1)
		{
			*hnext = hnext_s;
			return_value = e_RKQS_ERROR_SPIN_BOUNDS;
		}
		else if (return_value_xv==1 && return_value_s==1)
		{
			if ((hnext_xv<=0.0 && hnext_s<=0.0) || (hnext_xv>0.0 && hnext_s>0.0>=0.0))
			{
				return_value = e_RKQS_ERROR_COMBINED_BOUNDS;
				if (hnext_xv>0.0 && hnext_s>0.0) *hnext = fmin (hnext_xv, hnext_s);
				else if (hnext_xv<=0.0 && hnext_s<=0.0) *hnext = fmin (hnext_xv, hnext_s);
			}
			else return_value = e_RKQS_ERROR_REVERSED_INTERVAL;
			
		}
		else return_value = e_RKQS_ERROR_UNKNOWN;
	}
	
  /////////////////////////////////////////////////////////////////////////
	/////////////////// FINAL TIME STEP DECISION ////////////////////////////
	/////////////////////////////////////////////////////////////////////////
	
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HCURRENT, hcurrent);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EPSILON_MAX_XV, epsilon_max_xv);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EPSILON_MAX_S, epsilon_max_s);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_ERROR_XV, error_xv);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_ERROR_S, error_s);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT, *hnext);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_XV, hnext_xv);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_S, hnext_s);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_XV_PGROW_BOUNDED, hnext_xv_GB);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_S_PGROW_BOUNDED, hnext_s_GB);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_XV_PGROW_UNBOUNDED, hnext_xv_GU);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_S_PGROW_UNBOUNDED, hnext_s_GU);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_XV_PSHRNK_BOUNDED, hnext_xv_SB);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_S_PSHRNK_BOUNDED, hnext_s_SB);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_XV_PSHRNK_UNBOUNDED, hnext_xv_SU);
	// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_S_PSHRNK_UNBOUNDED, hnext_s_SU);
	// CUDA_RECORD_INT(d_IO_INT, vi_RECORD, e_d_IO_INT_RETURN_VALUE_XV, return_value_xv);
	// CUDA_RECORD_INT(d_IO_INT, vi_RECORD, e_d_IO_INT_RETURN_VALUE_S, return_value_s);
	// CUDA_RECORD_INT(d_IO_INT, vi_RECORD, e_d_IO_INT_RKQS_ERROR, return_value);
	return return_value;
}
__device__ /*__host__*/ int CUDA_RECORD_INT( 
	int *d_IO_INT, int vi_RECORD, int e_d_IO_INT_PARAM, int vi_PARAM)
{
	int vi_INT_IO_OFFSET = (getGlobalIdx_3D_3D()*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + vi_RECORD)*e_d_IO_INT_LAST;
	d_IO_INT[vi_INT_IO_OFFSET + e_d_IO_INT_PARAM] = vi_PARAM;
	return 0;
}
__device__ /*__host__*/ int CUDA_RECORD_DOUBLE( 
	double *d_IO,  int vi_RECORD, int e_d_IO_PARAM, double vd_PARAM)
{
	int vi_DOUBLE_IO_OFFSET = (getGlobalIdx_3D_3D()*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + vi_RECORD)*e_d_IO_LAST;
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_PARAM] = vd_PARAM;
	return 0;
}
__device__ /*__host__*/ int CUDA_RECORD_XVS( 
	double *d_IO, int *d_IO_INT,  int *p_vi_RECORD, 
	double l_time_CURRENT, double l_xvs[], double l_epsilon[], double l_xvs_scal[], double l_dxvsdt[], 
	double l_BField[], double l_dField_1D_FLAT[], double l_pol, int l_rkqs_TRIED, double l_hnext)
{
	int vi_DOUBLE_IO_OFFSET = (getGlobalIdx_3D_3D()*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + (*p_vi_RECORD))*e_d_IO_LAST;
	int vi_INT_IO_OFFSET = (getGlobalIdx_3D_3D()*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + (*p_vi_RECORD))*e_d_IO_INT_LAST;
	
	d_IO_INT[vi_INT_IO_OFFSET + e_d_IO_INT_ERROR] = 0;
	d_IO_INT[vi_INT_IO_OFFSET + e_d_IO_INT_THREAD] = getGlobalIdx_3D_3D();
	d_IO_INT[vi_INT_IO_OFFSET + e_d_IO_INT_RECORD] = (*p_vi_RECORD);
  d_IO_INT[vi_INT_IO_OFFSET + e_d_IO_INT_RKQS_STEPS] = l_rkqs_TRIED; 
	
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_T] = l_time_CURRENT;
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_X] = l_xvs[0];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_Y] = l_xvs[1];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_Z] = l_xvs[2];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_VX] = l_xvs[3];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_VY] = l_xvs[4];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_VZ] = l_xvs[5];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SX] = CUDA_SPIN_XVS_SINGLE(l_xvs, 0);
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SY] = CUDA_SPIN_XVS_SINGLE(l_xvs, 1);
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SZ] = CUDA_SPIN_XVS_SINGLE(l_xvs, 2);
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_X] = l_epsilon[0];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_Y] = l_epsilon[1];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_Z] = l_epsilon[2];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_VX] = l_epsilon[3];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_VY] = l_epsilon[4];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_VZ] = l_epsilon[5];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_X] = l_xvs_scal[0];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_Y] = l_xvs_scal[1];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_Z] = l_xvs_scal[2];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_VX] = l_xvs_scal[3];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_VY] = l_xvs_scal[4];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_VZ] = l_xvs_scal[5];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_RED_VX] = l_dxvsdt[0];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_RED_VY] = l_dxvsdt[1];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_RED_VZ] = l_dxvsdt[2];  
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_AX] = l_dxvsdt[3];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_AY] = l_dxvsdt[4];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_AZ] = l_dxvsdt[5];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_BX] = l_BField[0];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_BY] = l_BField[1];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_BZ] = l_BField[2];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_dB_XDX] = l_dField_1D_FLAT[0];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_dB_XDY] = l_dField_1D_FLAT[1];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_dB_XDZ] = l_dField_1D_FLAT[2];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_dB_YDX] = l_dField_1D_FLAT[3];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_dB_YDY] = l_dField_1D_FLAT[4];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_dB_YDZ] = l_dField_1D_FLAT[5];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_dB_ZDX] = l_dField_1D_FLAT[6];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_dB_ZDY] = l_dField_1D_FLAT[7];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_dB_ZDZ] = l_dField_1D_FLAT[8];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SPINNOR_0] = l_xvs[6];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SPINNOR_1] = l_xvs[7];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SPINNOR_2] = l_xvs[8];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SPINNOR_3] = l_xvs[9];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_DDT_SPINNOR_0] = l_dxvsdt[6]; 
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_DDT_SPINNOR_1] = l_dxvsdt[7]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_DDT_SPINNOR_2] = l_dxvsdt[8]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_DDT_SPINNOR_3] = l_dxvsdt[9]; 
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_SPINNOR_0] = l_epsilon[6]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_SPINNOR_1] = l_epsilon[7]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_SPINNOR_2] = l_epsilon[8]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_SPINNOR_3] = l_epsilon[9]; 	
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_SPINNOR_0] = l_xvs_scal[6]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_SPINNOR_1] = l_xvs_scal[7]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_SPINNOR_2] = l_xvs_scal[8]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_SPINNOR_3] = l_xvs_scal[9];
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_POLARIZATION] = l_pol;
	d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_HNEXT] = l_hnext;
	
	(*p_vi_RECORD)++;
	
	return 0;
}


__global__ void GENERIC_PIECEWISE_KERNEL_MULTI_XVS_RKQS_LOOP(
  double *d_IO, 
  int *d_IO_INT, 
	int numRecordsStart,
	int numRecordsEnd)
{
	int vi_RECORD_TEST_0, vi_RECORD_TEST_1, vi_RECORD_TEST_2, vi_RECORD_TEST_3,	vi_RECORD_TEST_4;
	int vi_RECORD_IO_OFFSET_END = 0, vi_IO_OFFSET_END, vi_IO_INT_OFFSET_END, vi_IO_DOUBLE_OFFSET_END;
	int vi_TESTOTESTO;
	int i, j, k, vi_RECORD, vi_RKQS_STEP, return_value_RKQS, l_rkqs_TRIED;
	int l_odeint_steps, vi_BREAK_FLAG, vi_REVERSE_FLAG, vi_INDEX;
	double l_time_CURRENT;
	double l_xvs[10];
	double l_dxvsdt[10];
	double l_xvs_scal[10];
	double l_epsilon[10];
	double l_BField[3];
	double l_dField_1D_FLAT[9];
	double l_pol;
	double l_hnext, l_hdid, l_htry;
	int vi_RECORD_IO_OFFSET_START = getGlobalIdx_3D_3D()*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + numRecordsStart;
	int vi_IO_OFFSET_START = vi_RECORD_IO_OFFSET_START*e_d_IO_LAST;
	int vi_IO_INT_OFFSET_START = vi_RECORD_IO_OFFSET_START*e_d_IO_INT_LAST;
	int vi_IO_DOUBLE_OFFSET_START = vi_RECORD_IO_OFFSET_START*e_d_IO_LAST;
	
	return_value_RKQS = d_IO[vi_IO_INT_OFFSET_START + e_d_IO_INT_ERROR];  
	l_rkqs_TRIED = 0;
	l_time_CURRENT = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_T]; 
	for (i = 0 ; i<10; i++)
	{
		l_xvs[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_X + i];
		l_dxvsdt[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_RED_VX + i];
		l_xvs_scal[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_SCAL_X + i];
		l_epsilon[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_ERR_X + i];
	}
	for (i = 0 ; i<3; i++)
	{
		l_BField[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_BX + i]; 
	}
	for (i = 0 ; i<9; i++)
	{
		l_dField_1D_FLAT[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_dB_XDX + i]; 
	}
	l_pol = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_POLARIZATION]; 
	l_hnext = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_HNEXT]; 
	
	double vd_TIME_START = l_time_CURRENT;
	
	for (i = 0 ; i<3; i++)
	{
		l_BField[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_BX + i]; 
		
	}
	for (i = 0 ; i<9; i++)
	{
		l_dField_1D_FLAT[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_dB_XDX + i]; 
		
	}
	l_pol = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_POLARIZATION]; 
	l_hnext = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_HNEXT]; 
	
	// CUDA_setspin( l_xvs, l_spin, l_spinnor, l_BField);
	int vi_CYCLE = 0;
	CUDA_derivs_XVS(l_time_CURRENT, l_xvs, l_dxvsdt, l_BField, l_dField_1D_FLAT);
	l_pol = CUDA_polcalc_XVS(l_BField[0], l_BField[1], l_BField[2], l_xvs);
	vi_REVERSE_FLAG = 0;
	// if (d_CONST[e_d_CONST_h1]<d_CONST[e_d_CONST_h1_SPIN]) l_htry = d_CONST[e_d_CONST_h1];
	// else l_htry = fmin(d_CONST[e_d_CONST_h1],d_CONST[e_d_CONST_h1_SPIN]);
	vi_RKQS_STEP = 0;
	return_value_RKQS = 0;
	while(vi_RECORD<numRecordsEnd)
	{
		l_htry = l_hnext;
		l_xvs_scal[0]=fabs(l_xvs[0])+fabs(l_dxvsdt[0]*l_htry)+d_CONST[e_d_CONST_def_TINY];
		l_xvs_scal[1]=fabs(l_xvs[1])+fabs(l_dxvsdt[1]*l_htry)+d_CONST[e_d_CONST_def_TINY];
		l_xvs_scal[2]=fabs(l_xvs[2])+fabs(l_dxvsdt[2]*l_htry)+d_CONST[e_d_CONST_def_TINY];
		l_xvs_scal[3]=fabs(l_xvs[3])+fabs(l_dxvsdt[3]*l_htry)+d_CONST[e_d_CONST_def_TINY];
		l_xvs_scal[4]=fabs(l_xvs[4])+fabs(l_dxvsdt[4]*l_htry)+d_CONST[e_d_CONST_def_TINY];
		l_xvs_scal[5]=fabs(l_xvs[5])+fabs(l_dxvsdt[5]*l_htry)+d_CONST[e_d_CONST_def_TINY];
		l_xvs_scal[6]=fabs(l_xvs[6])+fabs(l_dxvsdt[6]*l_htry)+d_CONST[e_d_CONST_def_TINY1];
		l_xvs_scal[7]=fabs(l_xvs[7])+fabs(l_dxvsdt[7]*l_htry)+d_CONST[e_d_CONST_def_TINY1];
		l_xvs_scal[8]=fabs(l_xvs[8])+fabs(l_dxvsdt[8]*l_htry)+d_CONST[e_d_CONST_def_TINY1];
		l_xvs_scal[9]=fabs(l_xvs[9])+fabs(l_dxvsdt[9]*l_htry)+d_CONST[e_d_CONST_def_TINY1];
		
		// if (0==0)/*(vi_RKQS_STEP>0 && (vi_RKQS_STEP%d_CONST_INT[e_d_CONST_INT_numCyclesPerRecord]))*/
			// {
				// if (d_CONST_INT[e_d_CONST_INT_numRecordsPerReverse]!=0)
				// {
					// if (d_CONST_INT[e_d_CONST_INT_numRecordsPerReverse_SPIN]==1 || 
						// (vi_RECORD%d_CONST_INT[e_d_CONST_INT_numRecordsPerReverse_SPIN])==0)
					// {
						// if (vi_REVERSE_FLAG==0) vi_REVERSE_FLAG = 1;
						// else vi_REVERSE_FLAG = 0;
					// }
				// }
				
			// }
		if (vi_RECORD==0)
		{
			CUDA_derivs_XVS(l_time_CURRENT, l_xvs, l_dxvsdt, l_BField, l_dField_1D_FLAT);
			l_pol = CUDA_polcalc_XVS(l_BField[0], l_BField[1], l_BField[2], l_xvs);
			CUDA_RECORD_XVS(d_IO, d_IO_INT,  &vi_RECORD, 
				l_time_CURRENT, l_xvs, l_epsilon, l_xvs_scal, l_dxvsdt, 
				l_BField, l_dField_1D_FLAT, l_pol, l_rkqs_TRIED, l_hnext);
		}
		else
		{
			return_value_RKQS = CUDA_rkqs_SINGLE_ATTEMPT_XVS(
				d_IO, 
				d_IO_INT, 
				vi_RECORD, 
				l_xvs, 
				l_dxvsdt, 
				&l_time_CURRENT, 
				l_htry, 
				&l_hdid, 
				&l_hnext, 
				l_xvs_scal, 
				l_BField, 
				l_dField_1D_FLAT, 
				l_epsilon, 
				&l_rkqs_TRIED);
			if (return_value_RKQS==e_RKQS_ERROR_NONE)
			{
				l_time_CURRENT = l_time_CURRENT + l_hdid;
				CUDA_derivs_XVS(l_time_CURRENT, l_xvs, l_dxvsdt, l_BField, l_dField_1D_FLAT);
				l_pol = CUDA_polcalc_XVS(l_BField[0], l_BField[1], l_BField[2], l_xvs);
				// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EXTRA_1, l_hnext);
				vi_RKQS_STEP++;
				if (vi_RKQS_STEP>=d_CONST_INT[e_d_CONST_INT_numCyclesPerRecord])
				{
					// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EXTRA_0, l_time_CURRENT);
					CUDA_RECORD_XVS(d_IO, d_IO_INT,  &vi_RECORD, 
						l_time_CURRENT, l_xvs, l_epsilon, l_xvs_scal, l_dxvsdt, 
						l_BField, l_dField_1D_FLAT, l_pol, l_rkqs_TRIED, l_hnext);
					vi_RKQS_STEP = 0;
				}
				l_rkqs_TRIED = 0;
			}
			else l_rkqs_TRIED++;
			
			// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EXTRA_1, l_hnext);
			// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EXTRA_2, l_htry);
			// CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EXTRA_3, l_hnext);
		}
	}
	
	// vi_RECORD_TEST_0 = vi_THREAD_IO_OFFSET;
	// vi_RECORD_TEST_1 = d_CONST_INT[e_d_CONST_INT_numRecordsPerThread];
	// vi_RECORD_TEST_2 = vi_THREAD_IO_OFFSET*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread];
	// vi_RECORD_TEST_3 = numRecordsEnd;
	// vi_RECORD_TEST_4 = vi_THREAD_IO_OFFSET*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + numRecordsEnd;
	
	int testsssss = getGlobalIdx_3D_3D();
	// vi_RECORD_IO_OFFSET_END = getGlobalIdx_3D_3D()*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + numRecordsEnd;
	// vi_IO_INT_OFFSET_END = vi_RECORD_IO_OFFSET_END*e_d_IO_INT_LAST;
	vi_IO_INT_OFFSET_END = (testsssss * d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + numRecordsEnd) * e_d_IO_INT_LAST;
	// vi_IO_DOUBLE_OFFSET_END = vi_RECORD_IO_OFFSET_END*e_d_IO_LAST;
	vi_IO_DOUBLE_OFFSET_END = (testsssss * d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + numRecordsEnd)*e_d_IO_LAST;

	// d_IO[vi_IO_INT_OFFSET_END + e_d_IO_INT_RKQS_ERROR] = vi_TESTOTESTO;  
	// d_IO[vi_IO_INT_OFFSET_END + e_d_IO_INT_RKQS_STEPS] = l_rkqs_TRIED;
	
	d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_T] = l_time_CURRENT; 
	for (i = 0 ; i<10; i++)
	{
		d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_X + i] = l_xvs[i];
		d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_RED_VX + i] = 	l_dxvsdt[i];
		d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_SCAL_X + i] = 	l_xvs_scal[i];
		d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_ERR_X + i] = 	l_epsilon[i];
	}
	for (i = 0 ; i<3; i++)
	{
		d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_BX + i] = l_BField[i]; 
	}
	for (i = 0 ; i<9; i++)
	{
		d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_dB_XDX + i] = l_dField_1D_FLAT[i]; 
	}
	d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_POLARIZATION] = l_pol; 
	d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_HNEXT] = l_hnext; 
	
	return;
}
void GENERIC_RECORD_FRAME(
	int param_numBlocks,
	int param_numThreadsPerBlock,
  double *d_IO,
  int *d_IO_INT,
	int numRecordsStart,
	int numRecordsEnd)
{
	// cudaPrintfInit ();
		// Run kernel
	hipDeviceSynchronize();
	GENERIC_PIECEWISE_KERNEL_MULTI_XVS_RKQS_LOOP<<< param_numBlocks, param_numThreadsPerBlock >>>(
		d_IO, 
		d_IO_INT, 
		numRecordsStart,
		numRecordsEnd);
	// GENERIC_PIECEWISE_KERNEL_MULTI_XVS<<<h_CONST_INT[e_d_CONST_INT_numBlocks],h_CONST_INT[e_d_CONST_INT_numThreadsPerBlock]>>>(
		// d_IO, 
		// d_IO_INT, 
		// vi_RecordsStartCurrent,
		// vi_RecordsEndCurrent);
	// GENERIC_PIECEWISE_KERNEL_MULTI<<<h_CONST_INT[e_d_CONST_INT_numBlocks],h_CONST_INT[e_d_CONST_INT_numThreadsPerBlock]>>>(
		// d_IO, 
		// d_IO_INT, 
		// vi_RecordsStartCurrent,
		// vi_RecordsEndCurrent);
	// cudaPrintfDisplay(stdout,true);
	// cudaPrintEnd();
	hipDeviceSynchronize();
	return;
}

void GENERIC_MIDDLEMAN_MULTI(
	const double *h_CONST,
	const int *h_CONST_INT,
  double *h_IO,
  int *h_IO_INT)
{
	printf("\n\n\nelloel,lleoeooo%d shoudl be zero\n\n",e_RKQS_ERROR_NONE);
	// Establish Scope parameters for simulation: 
	// number of neutrons, for how long, how many records to keep, etc.
	
	// Copy passsed constant values from Host Memory (CPU front side bus RAM) to Device Constant Memory.
	// Device Constant Memory is limited in size but accessible with close to register level latency at the the thread level due to mandatory caching in every CUDA Multi-processor.
  
	int vi_INDEX;
	
	double h_UNOFFICIAL_CONST[e_d_CONST_LAST];
	for (vi_INDEX = 0; vi_INDEX<e_d_CONST_LAST; vi_INDEX++) h_UNOFFICIAL_CONST[vi_INDEX] = h_CONST[vi_INDEX];
	int h_UNOFFICIAL_CONST_INT[e_d_CONST_INT_LAST];
	for (vi_INDEX = 0; vi_INDEX<e_d_CONST_INT_LAST; vi_INDEX++) h_UNOFFICIAL_CONST_INT[vi_INDEX] = h_CONST_INT[vi_INDEX];
	
	const int numBytesCONST = e_d_CONST_LAST*sizeof(double);
  const int numBytesCONST_INT = e_d_CONST_INT_LAST*sizeof(int);
  
	int vi_ERROR = hipMemcpyToSymbol(HIP_SYMBOL(d_CONST),h_UNOFFICIAL_CONST,numBytesCONST);
	int vi_ERROR_INT = hipMemcpyToSymbol(HIP_SYMBOL(d_CONST_INT),h_UNOFFICIAL_CONST_INT,numBytesCONST_INT);
  
	// Move passed input parameters specific to each thread from Host Memory to Device Memory (on-card RAM)
  double *d_IO = NULL;
  int *d_IO_INT = NULL;
  const int numBytesIO = h_CONST_INT[e_d_CONST_INT_numRecords]*e_d_IO_LAST*sizeof(double);
  const int numBytesIO_INT = h_CONST_INT[e_d_CONST_INT_numRecords]*e_d_IO_INT_LAST*sizeof(int);
  hipMalloc((void**)&d_IO, numBytesIO);
  hipMalloc((void**)&d_IO_INT, numBytesIO_INT);
  hipMemcpy(d_IO, h_IO, numBytesIO, hipMemcpyHostToDevice);
  hipMemcpy(d_IO_INT, h_IO_INT, numBytesIO_INT, hipMemcpyHostToDevice);
	
	printf("\ncheck eeeee check check");
	hipDeviceSynchronize();
	// Allocate where output data goes with room for all threads
  // double *d_OUT = NULL;
  // int *d_OUT_INT = NULL;
  // const int numBytesOUT = h_CONST_INT[e_d_CONST_INT_numRecords]*e_d_IO_LAST*sizeof(double);
  // const int numBytesOUT_INT = h_CONST_INT[e_d_CONST_INT_numRecords]*e_d_IO_INT_LAST*sizeof(int);
  // hipMalloc((void**)&d_OUT, numBytesOUT);
  // hipMalloc((void**)&d_OUT_INT, numBytesOUT_INT);
	int vi_RecordsStartCurrent = 0;
	int vi_RecordsEndCurrent = 0;
	int vi_RecordsEndFinal = h_CONST_INT[e_d_CONST_INT_numRecordsPerThread];
	int vi_UPDATES;
	for (vi_UPDATES = 0; vi_RecordsEndCurrent<vi_RecordsEndFinal; vi_UPDATES++)
	{
		// Run kernel
		vi_RecordsStartCurrent = vi_RecordsEndCurrent;
		int vi_RecordsEndCurrent_TEMP =  vi_RecordsStartCurrent + h_CONST_INT[e_d_CONST_INT_numRecordsPerUpdate];
		if (vi_RecordsEndCurrent_TEMP>vi_RecordsEndFinal) vi_RecordsEndCurrent = vi_RecordsEndFinal;
		else vi_RecordsEndCurrent = vi_RecordsStartCurrent + h_CONST_INT[e_d_CONST_INT_numRecordsPerUpdate];
		// vi_RecordsStartCurrent = 0;
		// vi_RecordsEndCurrent = vi_RecordsEndFinal;
		printf("\nStarting Records %d-%d of %d... ",vi_RecordsStartCurrent,vi_RecordsEndCurrent,h_CONST_INT[e_d_CONST_INT_numRecordsPerThread]);
		GENERIC_RECORD_FRAME(
			h_CONST_INT[e_d_CONST_INT_numBlocks],
			h_CONST_INT[e_d_CONST_INT_numThreadsPerBlock],
			d_IO,
			d_IO_INT,
			vi_RecordsStartCurrent,
			vi_RecordsEndCurrent);
		// GENERIC_PIECEWISE_KERNEL_MULTI<<<h_CONST_INT[e_d_CONST_INT_numBlocks],h_CONST_INT[e_d_CONST_INT_numThreadsPerBlock]>>>(
			// d_IO, 
			// d_IO_INT, 
			// vi_RecordsStartCurrent,
			// vi_RecordsEndCurrent);
		printf("Completed");
	}
  printf("\nDay o day o daylight come and me wanna go %d threads",h_CONST_INT[e_d_CONST_INT_numThreads]);
	// Move results of output to Host Memory from Device Memory
  hipMemcpy(h_IO, d_IO, numBytesIO, hipMemcpyDeviceToHost);
  hipMemcpy(h_IO_INT, d_IO_INT, numBytesIO_INT, hipMemcpyDeviceToHost);
  
  hipFree(d_IO);
  hipFree(d_IO_INT);
  
	return;
}
