
/* Include files ----------------------------------------------------------------- */

#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <iostream>
#include <ctime>
#include <iomanip>
#include <cmath>
#include <limits>
#include <string>

#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include "UCN_CUDA_ALL_KERNEL.cuh"
#include "UCN_CUDA_WRAPPER.h"

int GENERIC_WRAPPER_MULTI(
	const double *h_CONST,
	const int *h_CONST_INT,
  double *h_IO,
  int *h_IO_INT)
{
	GENERIC_MIDDLEMAN_MULTI(
		h_CONST,
		h_CONST_INT,
		h_IO,
		h_IO_INT);
	return 0;
}