#include "hip/hip_runtime.h"

/* Include files ----------------------------------------------------------------- */
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// #include "cuPrintf.cuh"
// #include "cuPrintf.cu"

/* 

    ###########################################################
    # This is version 2.0
    # CUDA Runge-Kutta 4th/5th Order
    #
    # Author : Spencer McBride Day Moore
    # Date   : May 15, 2015
    ###########################################################

    This is a standalone nvcc compiled program that implements a function in order
    to be implemented with double precision floating point numbers compute capability
    of CUDA device must be 2.0 or higher (-arch=sm_20 is the flag that goes with nvcc 
    for 2.0, -arch=sm_21 for 2.1 and so on). the #define variables are listed below 
    the header files.  Also the function fxn_dydx is the example given is only allowed
    single variable dependence in addition to time f_x(t,x)=dx/dt, f_y(t,y)=dy/dt etc. 
    and not f_x(t,x,y,z)=dx/dt etc..  This will need to be changed for usefulness in 
    the neutron transport code.  Currently the user can pass variables to the 
    individual threads via the constant double arrays cu_A and cu_B.  They are currently
    initialized ot random values between positive and negative one for testing purposes. 
    There is no adaptive step implemented although the stepwise error is outputted along
    with (t,x,y,z).
*/

#include "UCN_CUDA_ALL_KERNEL.cuh"

__constant__ int d_CONST_INT[e_d_CONST_INT_LAST];
__constant__ double d_CONST[e_d_CONST_LAST];

__device__ int CUDA_ipow( int base, int exp)
{
  int result = -1;
  if (exp<0) result = 0;
  else if (exp==0) result = 1;
  else if (exp>1)
  {
    result = 1;
    for (int i = exp; i>0; i--)
    {
        result *= base;
    }
  }
  return result;
}
__device__ int getGlobal_blockId_3D()
{
  int blockId = blockIdx.x + 
    (blockIdx.y * gridDim.x)  + (
    gridDim.x * gridDim.y * blockIdx.z);
  return blockId;
}
__device__ int getGlobalIdx_3D_3D()
{
  int blockId = getGlobal_blockId_3D();
  int threadId = 
    (blockId * blockDim.x * blockDim.y * blockDim.z) + 
    threadIdx.z * (blockDim.x * blockDim.y) + 
    threadIdx.y * blockDim.x + 
    threadIdx.x;
  return threadId;
  // int vi_RETURN_VALUE = ( blockIdx.x * blockDim.x + threadIdx.x );
  // return vi_RETURN_VALUE
}
__device__ int fvi_ISOLATE_INT_RANGE( int vi_INPUT, int vi_MSB, int vi_LSB)
{
  int vi_ABS_INPUT = vi_INPUT < 0 ? -vi_INPUT : vi_INPUT;
  int vi_INDEX, vi_MOD;
  for (vi_INDEX = 1, vi_MOD = 1 ; vi_INDEX<vi_MSB; vi_INDEX++)
  {
    vi_MOD = vi_MOD * 2;
  }
  int vi_INTERMEDIATE = ( vi_ABS_INPUT % vi_MOD );
  int vi_FINAL = ( vi_INTERMEDIATE >> ( vi_LSB - 1 ) );
  if ( vi_FINAL != ( vi_INTERMEDIATE * CUDA_ipow(2, vi_LSB - 1))) vi_FINAL = -1;
  return vi_FINAL;
}
__device__ /*__host__*/ double CUDA_theta( double x, double y, double z)
{
  double temp;
  if ( (z!=0) && ((x*x+y*y)!=0) ) temp = atan2(sqrt(x*x+y*y),z);
  else if (z==0) temp = d_CONST[e_d_CONST_def_PI]/2.0;
  else temp = d_CONST[e_d_CONST_def_TINY];  
  if (temp >= d_CONST[e_d_CONST_def_PI]) temp=2*d_CONST[e_d_CONST_def_PI]-temp;  
  return(temp);
}
__device__ /*__host__*/ double CUDA_ro( double x, double y)
{
  double solution;
  if (x*x+y*y != 0) solution = sqrt(x*x + y*y);
  else solution = ( d_CONST[e_d_CONST_def_TINY] );
  return solution;
}
__device__ /*__host__*/ double CUDA_r( double x, double y, double z)
{
  double solution;
  if ((x*x+y*y+z*z)!= 0) solution = (sqrt(x*x+y*y+z*z));
  else solution = ( d_CONST[e_d_CONST_def_TINY] );
  return solution;
}
__device__ /*__host__*/ double CUDA_phi( double x, double y)
{
  double solution; 
  if (x!=0)
    solution = atan2(y,x);
  else
    solution = ((d_CONST[e_d_CONST_def_PI]/2.0));
  return solution;
}
__device__ /*__host__*/ double CUDA_polcalc_XVS( 
  double bb1x, 
  double bb1y, 
  double bb1z, 
  double spinor[])
{
  double xhat1,yhat1,zhat1,snorm;
  /* const double xhat2,yhat2,zhat2; */
  double bxhat,byhat,bzhat;
  double bnorm,polarization;
  /* const double spinnorm,rnorm,rtest; */

  bnorm = sqrt(bb1x*bb1x + bb1y*bb1y + bb1z*bb1z);
  bxhat = bb1x/bnorm;
  byhat = bb1y/bnorm;
  bzhat = bb1z/bnorm;
  
  xhat1 = 0.;
  yhat1 = 0.;
  zhat1 = 0.;
  // CUDA_SPIN_XVS_XYZ( spin, spinor );
  snorm = (spinor[0]*spinor[0]) + (spinor[1]*spinor[1]) +  
          (spinor[2]*spinor[2]) + (spinor[3]*spinor[3]);
  xhat1 =( 2.*((spinor[0]*spinor[2]) + (spinor[1]*spinor[3])))/snorm;
  yhat1 = ( 2.*((spinor[0]*spinor[3]) - (spinor[1]*spinor[2])))/snorm;
  zhat1 = ((spinor[0]*spinor[0]) + (spinor[1]*spinor[1]) - 
     (spinor[2]*spinor[2]) - (spinor[3]*spinor[3]))/snorm;
  polarization = xhat1*bxhat + yhat1*byhat + zhat1*bzhat;
  return(polarization);
}
__device__ /*__host__*/ int CUDA_derivs_XVS( 
  double t,
  double spinor[],
  double dspinordt[],
  double BField[])
{  
  double rfstr = d_CONST[e_d_CONST_RF_BFIELD_MAG];
  double  brf[4],omega,btoomega,zrf,zdist,zdelta;
  btoomega = d_CONST[e_d_CONST_def_MOMENT]/d_CONST[e_d_CONST_def_HBAR];  // coeff. for psi-dot 
  omega    = 2.0*d_CONST[e_d_CONST_def_MOMENT]*(1.0)/d_CONST[e_d_CONST_def_HBAR]; // reson. frequ. for 1.0 T field 
  zrf      = 1.14; // chosen to match crossing poconst int 
  // some comment explaining this. v 
	double t_INTERVAL = inter[e_inter_t_FINAL] - inter[e_inter_t_INITIAL];
	double t_STEP = t - inter[e_inter_t_INITIAL];
	double BField_STEP[3];
	spin_INTERVAL_3 = inter[e_inter_spin_FINAL_3] - inter[e_inter_spin_INITIAL_3];
  double BField_INTERVAL_X = inter[e_inter_BField_FINAL_X] - inter[e_inter_BField_INITIAL_X];
  double BField_INTERVAL_Y = inter[e_inter_BField_FINAL_Y] - inter[e_inter_BField_INITIAL_Y];
  double BField_INTERVAL_Z = inter[e_inter_BField_FINAL_Z] - inter[e_inter_BField_INITIAL_Z];
  BField[0] = (BField_INTERVAL_X/t_INTERVAL)*t_STEP + inter[e_inter_BField_INITIAL_X];
  BField[1] = (BField_INTERVAL_Y/t_INTERVAL)*t_STEP + inter[e_inter_BField_INITIAL_Y];
  BField[2] = (BField_INTERVAL_Z/t_INTERVAL)*t_STEP + inter[e_inter_BField_INITIAL_Z];
  
	zdist = (spin_INTERVAL_3/t_INTERVAL)*t_STEP+inter[e_inter_spin_INITIAL_3]; //spin[2];
  zdelta = (zdist - zrf)*(zdist -zrf)/(.05*.05);
  brf[0] = rfstr*cos(omega*(t-0.0)) * (1.0/((1.0+zdelta)*(1.0+zdelta))); 
  brf[1] = 0.;
    // exp(-(zdist-zrf)*(zdist-zrf)/(.05*.05)); 
  brf[2] = 0.;
  int i;
  double extra_brf[3];
  // CUDA_RF_BFIELD(t, spinor, extra_brf);
  if (d_CONST_INT[e_d_CONST_INT_FLAG_RF]==1)
  {
    for ( i = 0; i<3; i++) BField[i] += brf[i]; // extra_brf[i];
  }
  
  // Classical Description of Spin 
  // NB moment has "I=1/2" already... 
  // dspinordt[0] = 0.0;
  // dspinordt[1] = btoomega*(spinor[2]*BField[2] - spinor[3]*BField[1]);
  // dspinordt[2] = btoomega*(spinor[3]*BField[0] - spinor[1]*BField[2]);
  // dspinordt[3] = btoomega*(spinor[1]*BField[1] - spinor[2]*BField[0]);
  // ihd/dt = (moment)*(sigma)*B 
  // NB moment has "I=1/2" already... 
  dspinordt[0] = btoomega * ( (BField[2] * spinor[1]) + (BField[0] * spinor[3]) - (BField[1] * spinor[2]));
  dspinordt[1] = btoomega * (-(BField[2] * spinor[0]) - (BField[0] * spinor[2]) - (BField[1] * spinor[3]));
  dspinordt[2] = btoomega * (-(BField[2] * spinor[3]) + (BField[0] * spinor[1]) + (BField[1] * spinor[0]));
  dspinordt[3] = btoomega * ( (BField[2] * spinor[2]) - (BField[0] * spinor[0]) + (BField[1] * spinor[1]));
  return 0;
}
__device__ /*__host__*/ int CUDA_rkck_XVS( 
  double *d_IO, 
  int *d_IO_INT, 
  int vi_RECORD,
  double t,
  double spinor[],
  double dspinordt[],
  double h,
  double spinorerr[],
  double spinorout[],
  double BField[])
{
  // double BField[3][3];
  CUDA_derivs_XVS(t, spinor, dspinordt, BField);
  int i;
  /*   static const a2=(0.2, a3=(0.3, a4=(0.6, a5=(1.0, a6=(0.875; */
  const double b21=(0.2), b31=(3.0/40.0), b32=(9.0/40.0), b41=(0.3), b42=( -0.9), b43=(1.2);
  const double b51=( -11.0/54.0), b52=(2.5), b53=( -70.0/27.0), b54=(35.0/27.0); 
  const double b61=(1631.0/55296.0), b62=(175.0/512.0), b63=(575.0/13824.0), b64=(44275.0/110592.0), b65=(253.0/4096.0); 
  const double c1=(37.0/378.0), c3=(250.0/621.0), c4=(125.0/594.0), c6=(512.0/1771.0);
  const double dc5=( -277.0/14336.0), dc1=(c1-2825.0/27648.0),  dc3=(c3-18575.0/48384.0), dc4=(c4-13525.0/55296.0), dc6=(c6-0.25);
  double ak2[4], ak3[4], ak4[4], ak5[4], ak6[4], spinortemp[4];
  for (i = 0; i<4; i++)   /* First step */ spinortemp[i]=spinor[i]+b21*h*dspinordt[i];
  CUDA_derivs_XVS(t, spinor, ak2, BField);    /* Second step */
  for (i = 0; i<4; i++) spinortemp[i]=spinor[i]+h*(b31*dspinordt[i]+b32*ak2[i]);
  CUDA_derivs_XVS(t, spinor, ak3, BField);    /* Third step */
  for (i = 0; i<4; i++) spinortemp[i]=spinor[i]+h*(b41*dspinordt[i]+b42*ak2[i]+b43*ak3[i]);
  CUDA_derivs_XVS(t, spinor, ak4, BField);    /* Fourth step */
  for (i = 0; i<4; i++) spinortemp[i]=spinor[i]+h*(b51*dspinordt[i]+b52*ak2[i]+b53*ak3[i]+b54*ak4[i]);
  CUDA_derivs_XVS(t, spinortemp, ak5, BField);    /* Fifth step */
  for (i = 0; i<4; i++) spinortemp[i]=spinor[i]+h*(b61*dspinordt[i]+b62*ak2[i]+b63*ak3[i]+b64*ak4[i]+b65*ak5[i]);
  CUDA_derivs_XVS(t, spinor, ak6, BField);    /* Sixth step */
  for (i = 0; i<4; i++)  spinorout[i]=spinor[i]+h*(c1*dspinordt[i]+c3*ak3[i]+c4*ak4[i]+c6*ak6[i]); /* Accumulate increments with proper weights */
  for (i = 0; i<4; i++) spinorerr[i]=h*(dc1*dspinordt[i]+dc3*ak3[i]+dc4*ak4[i]+dc5*ak5[i]+dc6*ak6[i]);
  return 0;
}
__device__ /*__host__*/ int CUDA_rkqs_SINGLE_ATTEMPT_XVS(
  double *d_IO, 
  int *d_IO_INT,
  int vi_RECORD, 
  double spinor[],
   double dspinordt[],
   double *t,
   double htry,
  double *hdid,
   double *hnext,
   double spinor_scal[],
   double BField[],
   double epsilon[],
   int *rkqs_TRIED,
	 double inter[])
{
  int i, j, k;
  double hnext_xv;
  double hnext_s;
  double hcurrent = htry;
  double spinor_temp[4];
  double epsilon_temp[4];
  double error, error_xv, error_s;
  int return_value = -1;
  int return_value_xv = -1;
  int return_value_s = -1;
  double abs_max_xv, abs_max_s;
  double hnext_s_GB, hnext_s_GU, hnext_s_SB, hnext_s_SU;
  double hnext_xv_GB, hnext_xv_GU, hnext_xv_SB, hnext_xv_SU;
  CUDA_rkck_XVS( 
    d_IO, 
    d_IO_INT, 
    vi_RECORD, 
    *t, 
    spinor, 
    dspinordt, 
    hcurrent, 
    epsilon_temp, 
    spinor_temp, 
    BField);
  double epsilon_max_xv = 0.0;
  double epsilon_max_s = 0.0;
  for (i = 0; i<4; i++)
  {
		abs_max_s = fabs(epsilon_temp[i]/spinor_scal[i]);
		// if (abs_max_s<0) abs_max_s *= -1.0; 
		if (abs_max_s>epsilon_max_s) epsilon_max_s = abs_max_s;
  }
  error_s = epsilon_max_s/d_CONST[e_d_CONST_def_MAXERR1];
  
  /////////////////////////////////////////////////////////////////////////
  /////////////////// SPIN STEP COMPUTATION ////////////////////////
  /////////////////////////////////////////////////////////////////////////
  
  /* S: Step succeeded. Compute size of next step */
  hnext_s_GU = d_CONST[e_d_CONST_def_SAFETY1]*hcurrent*pow(error_s,d_CONST[e_d_CONST_def_PGROW1]);
  /* S: No more than a factor of 5 increase */
  hnext_s_GB = 5.0*hcurrent; // ((hnext_s_GU >= 0.0) ? fmin(hnext_s_GU, 5.0*hcurrent) : fmax(hnext_s_GU, 5.0*hcurrent));
  /* S: Truncation error too large. Reduce stepsize */
  hnext_s_SU = d_CONST[e_d_CONST_def_SAFETY1]*hcurrent*pow(error_s, d_CONST[e_d_CONST_def_PSHRNK1]);
  /* S: No more than a factor of 10 */
  hnext_s_SB = 0.1*hcurrent; // ((hcurrent >= 0.0) ? fmax(hnext_s_SU, 0.1*hcurrent) : fmin(hnext_s_SU, 0.1*hcurrent));
  
 /////////////////////////////////////////////////////////////////////////
  ////////////////// SPIN ERROR AND STEP DECISION /////////////////////////
  /////////////////////////////////////////////////////////////////////////
  
  if (error_s <= 1.0)
  {
    hnext_s = ((hcurrent >= 0.0) ? fmin(hnext_s_GU, hnext_s_GB) : fmax(hnext_s_GU, hnext_s_GB));
    return_value_s = 0;
  }
  else if (error_s > 1.0)
  {
    hnext_s = ((hcurrent >= 0.0) ? fmax(hnext_s_SU, hnext_s_SB) : fmin(hnext_s_SU, hnext_s_SB));
    return_value_s = 1;
  }
  
  /////////////////////////////////////////////////////////////////////////
  /////////////////// FINAL TIME STEP SIGN CHECK //////////////////////////
  /////////////////////////////////////////////////////////////////////////
  
  
  
  /////////////////////////////////////////////////////////////////////////
  /////////////////// FINAL TIME STEP DECISION ////////////////////////////
  /////////////////////////////////////////////////////////////////////////
  
  /* nrerror("stepsize underflow in rkqs"); */ /* diag */
  if ((*t + hcurrent) == *t) return_value = e_RKQS_ERROR_STEPSIZE_UNDERFLOW;
  else if (return_value_xv==0 && return_value_s==0)
  {
    for (i=0; i<4; i++)
    {
      spinor[i] = spinor_temp[i];
      epsilon[i] = epsilon_temp[i];
    }
    *t += hcurrent;
    *hdid = hcurrent;
    *rkqs_TRIED = 0;
    if (hcurrent<=0.0)
    {
      *hnext = fmax ( hnext_s , fmax ( d_CONST[e_d_CONST_tframe], d_CONST[e_d_CONST_tframe_SPIN]));
    }
    else if (hcurrent>0.0) 
    {
      *hnext = fmin ( hnext_s , fmin ( d_CONST[e_d_CONST_tframe], d_CONST[e_d_CONST_tframe_SPIN]));
    }
    return_value = e_RKQS_ERROR_NONE;
  }
  else
  {
    *hdid = 0.0;
    (*rkqs_TRIED)++;
    if (return_value_s==1)
    {
      *hnext = hnext_s;
      return_value = e_RKQS_ERROR_SPIN_BOUNDS;
    }
    else return_value = e_RKQS_ERROR_UNKNOWN;
  }
  
  /////////////////////////////////////////////////////////////////////////
  /////////////////// FINAL TIME STEP DECISION ////////////////////////////
  /////////////////////////////////////////////////////////////////////////
  
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HCURRENT, hcurrent);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EPSILON_MAX_XV, epsilon_max_xv);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EPSILON_MAX_S, epsilon_max_s);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_ERROR_XV, error_xv);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_ERROR_S, error_s);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT, *hnext);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_XV, hnext_xv);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_S, hnext_s);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_XV_PGROW_BOUNDED, hnext_xv_GB);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_S_PGROW_BOUNDED, hnext_s_GB);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_XV_PGROW_UNBOUNDED, hnext_xv_GU);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_S_PGROW_UNBOUNDED, hnext_s_GU);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_XV_PSHRNK_BOUNDED, hnext_xv_SB);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_S_PSHRNK_BOUNDED, hnext_s_SB);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_XV_PSHRNK_UNBOUNDED, hnext_xv_SU);
  // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_HNEXT_S_PSHRNK_UNBOUNDED, hnext_s_SU);
  // CUDA_RECORD_INT(d_IO_INT, vi_RECORD, e_d_IO_INT_RETURN_VALUE_XV, return_value_xv);
  // CUDA_RECORD_INT(d_IO_INT, vi_RECORD, e_d_IO_INT_RETURN_VALUE_S, return_value_s);
  // CUDA_RECORD_INT(d_IO_INT, vi_RECORD, e_d_IO_INT_RKQS_ERROR, return_value);
  return return_value;
}
__device__ /*__host__*/ int CUDA_RECORD_INT( 
  int *d_IO_INT, int vi_RECORD, int e_d_IO_INT_PARAM, int vi_PARAM)
{
  int vi_INT_IO_OFFSET = (getGlobalIdx_3D_3D()*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + vi_RECORD)*e_d_IO_INT_LAST;
  d_IO_INT[vi_INT_IO_OFFSET + e_d_IO_INT_PARAM] = vi_PARAM;
  return 0;
}
__device__ /*__host__*/ int CUDA_RECORD_DOUBLE( 
  double *d_IO,  int vi_RECORD, int e_d_IO_PARAM, double vd_PARAM)
{
  int vi_DOUBLE_IO_OFFSET = (getGlobalIdx_3D_3D()*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + vi_RECORD)*e_d_IO_LAST;
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_PARAM] = vd_PARAM;
  return 0;
}
__device__ /*__host__*/ int CUDA_RECORD_XVS( 
  double *d_IO, int *d_IO_INT,  int *p_vi_RECORD, 
  double l_time_CURRENT, double l_spinor[], double l_epsilon[], double l_spinor_scal[], double l_dspinordt[], 
  double l_BField[], double l_pol, int l_rkqs_TRIED, double l_hnext)
{
  int vi_DOUBLE_IO_OFFSET = (getGlobalIdx_3D_3D()*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + (*p_vi_RECORD))*e_d_IO_LAST;
  int vi_INT_IO_OFFSET = (getGlobalIdx_3D_3D()*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + (*p_vi_RECORD))*e_d_IO_INT_LAST;
  
  d_IO_INT[vi_INT_IO_OFFSET + e_d_IO_INT_ERROR] = 0;
  d_IO_INT[vi_INT_IO_OFFSET + e_d_IO_INT_THREAD] = getGlobalIdx_3D_3D();
  d_IO_INT[vi_INT_IO_OFFSET + e_d_IO_INT_RECORD] = (*p_vi_RECORD);
  d_IO_INT[vi_INT_IO_OFFSET + e_d_IO_INT_RKQS_STEPS] = l_rkqs_TRIED; 
  
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_T] = l_time_CURRENT;
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_BX] = l_BField[0];
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_BY] = l_BField[1];
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_BZ] = l_BField[2];
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SPINNOR_0] = l_spinor[0];
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SPINNOR_1] = l_spinor[1];
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SPINNOR_2] = l_spinor[2];
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SPINNOR_3] = l_spinor[3];
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_DDT_SPINNOR_0] = l_dspinordt[0]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_DDT_SPINNOR_1] = l_dspinordt[1]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_DDT_SPINNOR_2] = l_dspinordt[2]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_DDT_SPINNOR_3] = l_dspinordt[3]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_SPINNOR_0] = l_epsilon[0]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_SPINNOR_1] = l_epsilon[1]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_SPINNOR_2] = l_epsilon[2]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_ERR_SPINNOR_3] = l_epsilon[3];   
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_SPINNOR_0] = l_spinor_scal[0]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_SPINNOR_1] = l_spinor_scal[1]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_SPINNOR_2] = l_spinor_scal[2]; 
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_SCAL_SPINNOR_3] = l_spinor_scal[3];
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_POLARIZATION] = l_pol;
  d_IO[vi_DOUBLE_IO_OFFSET + e_d_IO_HNEXT] = l_hnext;
  
  (*p_vi_RECORD)++;
  
  return 0;
}


__global__ void GENERIC_PIECEWISE_KERNEL_MULTI_XVS_RKQS_LOOP(
  double *d_IO, 
  int *d_IO_INT, 
  int numRecordsStart,
  int numRecordsEnd)
{
  int vi_RECORD_TEST_0, vi_RECORD_TEST_1, vi_RECORD_TEST_2, vi_RECORD_TEST_3,  vi_RECORD_TEST_4;
  int vi_RECORD_IO_OFFSET_END = 0, vi_IO_OFFSET_END, vi_IO_INT_OFFSET_END, vi_IO_DOUBLE_OFFSET_END;
  int vi_TESTOTESTO;
  int i, j, k, vi_RECORD, vi_RKQS_STEP, return_value_RKQS, l_rkqs_TRIED;
  int l_odeint_steps, vi_BREAK_FLAG, vi_REVERSE_FLAG, vi_INDEX;
  double l_time_CURRENT;
  double l_spinor[4];
  double l_dspinordt[4];
  double l_spinor_scal[4];
  double l_epsilon[4];
  double l_BField[3];
  double l_pol;
  double l_hnext, l_hdid, l_htry;
  int vi_RECORD_IO_OFFSET_START = getGlobalIdx_3D_3D()*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + numRecordsStart;
  int vi_IO_OFFSET_START = vi_RECORD_IO_OFFSET_START*e_d_IO_LAST;
  int vi_IO_INT_OFFSET_START = vi_RECORD_IO_OFFSET_START*e_d_IO_INT_LAST;
  int vi_IO_DOUBLE_OFFSET_START = vi_RECORD_IO_OFFSET_START*e_d_IO_LAST;
  
  return_value_RKQS = d_IO[vi_IO_INT_OFFSET_START + e_d_IO_INT_ERROR];  
  l_rkqs_TRIED = 0;
  l_time_CURRENT = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_T]; 
  for (i = 0 ; i<4; i++)
  {
    l_spinor[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_X + i];
    l_dspinordt[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_RED_VX + i];
    l_spinor_scal[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_SCAL_X + i];
    l_epsilon[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_ERR_X + i];
  }
  for (i = 0 ; i<3; i++)
  {
    l_BField[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_BX + i]; 
  }
  l_pol = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_POLARIZATION]; 
  l_hnext = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_HNEXT]; 
  
  double vd_TIME_START = l_time_CURRENT;
  
  for (i = 0 ; i<3; i++)
  {
    l_BField[i] = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_BX + i]; 
    
  }
  l_pol = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_POLARIZATION]; 
  l_hnext = d_IO[vi_IO_DOUBLE_OFFSET_START + e_d_IO_HNEXT]; 
  
  // CUDA_setspin( l_spinor, l_spin, l_spinnor, l_BField);
  int vi_CYCLE = 0;
  CUDA_derivs_XVS(l_time_CURRENT, l_spinor, l_dspinordt, l_BField);
  l_pol = CUDA_polcalc_XVS(l_BField[0], l_BField[1], l_BField[2], l_spinor);
  vi_REVERSE_FLAG = 0;
  // if (d_CONST[e_d_CONST_h1]<d_CONST[e_d_CONST_h1_SPIN]) l_htry = d_CONST[e_d_CONST_h1];
  // else l_htry = fmin(d_CONST[e_d_CONST_h1],d_CONST[e_d_CONST_h1_SPIN]);
  vi_RKQS_STEP = 0;
  return_value_RKQS = 0;
  while(vi_RECORD<numRecordsEnd)
  {
    l_htry = l_hnext;
    l_spinor_scal[0]=fabs(l_spinor[0])+fabs(l_dspinordt[0]*l_htry)+d_CONST[e_d_CONST_def_TINY1];
    l_spinor_scal[1]=fabs(l_spinor[1])+fabs(l_dspinordt[1]*l_htry)+d_CONST[e_d_CONST_def_TINY1];
    l_spinor_scal[2]=fabs(l_spinor[2])+fabs(l_dspinordt[2]*l_htry)+d_CONST[e_d_CONST_def_TINY1];
    l_spinor_scal[3]=fabs(l_spinor[3])+fabs(l_dspinordt[3]*l_htry)+d_CONST[e_d_CONST_def_TINY1];
    
    // if (0==0)/*(vi_RKQS_STEP>0 && (vi_RKQS_STEP%d_CONST_INT[e_d_CONST_INT_numCyclesPerRecord]))*/
      // {
        // if (d_CONST_INT[e_d_CONST_INT_numRecordsPerReverse]!=0)
        // {
          // if (d_CONST_INT[e_d_CONST_INT_numRecordsPerReverse_SPIN]==1 || 
            // (vi_RECORD%d_CONST_INT[e_d_CONST_INT_numRecordsPerReverse_SPIN])==0)
          // {
            // if (vi_REVERSE_FLAG==0) vi_REVERSE_FLAG = 1;
            // else vi_REVERSE_FLAG = 0;
          // }
        // }
        
      // }
    if (vi_RECORD==0)
    {
      CUDA_derivs_XVS(l_time_CURRENT, l_spinor, l_dspinordt, l_BField);
      l_pol = CUDA_polcalc_XVS(l_BField[0], l_BField[1], l_BField[2], l_spinor);
      CUDA_RECORD_XVS(d_IO, d_IO_INT,  &vi_RECORD, 
        l_time_CURRENT, l_spinor, l_epsilon, l_spinor_scal, l_dspinordt, 
        l_BField, l_pol, l_rkqs_TRIED, l_hnext);
    }
    else
    {
      return_value_RKQS = CUDA_rkqs_SINGLE_ATTEMPT_XVS(
        d_IO, 
        d_IO_INT, 
        vi_RECORD, 
        l_spinor, 
        l_dspinordt, 
        &l_time_CURRENT, 
        l_htry, 
        &l_hdid, 
        &l_hnext, 
        l_spinor_scal, 
        l_BField, 
        l_epsilon, 
        &l_rkqs_TRIED);
      if (return_value_RKQS==e_RKQS_ERROR_NONE)
      {
        l_time_CURRENT = l_time_CURRENT + l_hdid;
        CUDA_derivs_XVS(l_time_CURRENT, l_spinor, l_dspinordt, l_BField);
        l_pol = CUDA_polcalc_XVS(l_BField[0], l_BField[1], l_BField[2], l_spinor);
        // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EXTRA_1, l_hnext);
        vi_RKQS_STEP++;
        if (vi_RKQS_STEP>=d_CONST_INT[e_d_CONST_INT_numCyclesPerRecord])
        {
          // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EXTRA_0, l_time_CURRENT);
          CUDA_RECORD_XVS(d_IO, d_IO_INT,  &vi_RECORD, 
            l_time_CURRENT, l_spinor, l_epsilon, l_spinor_scal, l_dspinordt, 
            l_BField, l_pol, l_rkqs_TRIED, l_hnext);
          vi_RKQS_STEP = 0;
        }
        l_rkqs_TRIED = 0;
      }
      else l_rkqs_TRIED++;
      
      // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EXTRA_1, l_hnext);
      // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EXTRA_2, l_htry);
      // CUDA_RECORD_DOUBLE(d_IO, vi_RECORD, e_d_IO_EXTRA_3, l_hnext);
    }
  }
  
  // vi_RECORD_TEST_0 = vi_THREAD_IO_OFFSET;
  // vi_RECORD_TEST_1 = d_CONST_INT[e_d_CONST_INT_numRecordsPerThread];
  // vi_RECORD_TEST_2 = vi_THREAD_IO_OFFSET*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread];
  // vi_RECORD_TEST_3 = numRecordsEnd;
  // vi_RECORD_TEST_4 = vi_THREAD_IO_OFFSET*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + numRecordsEnd;
  
  int testsssss = getGlobalIdx_3D_3D();
  // vi_RECORD_IO_OFFSET_END = getGlobalIdx_3D_3D()*d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + numRecordsEnd;
  // vi_IO_INT_OFFSET_END = vi_RECORD_IO_OFFSET_END*e_d_IO_INT_LAST;
  vi_IO_INT_OFFSET_END = (testsssss * d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + numRecordsEnd) * e_d_IO_INT_LAST;
  // vi_IO_DOUBLE_OFFSET_END = vi_RECORD_IO_OFFSET_END*e_d_IO_LAST;
  vi_IO_DOUBLE_OFFSET_END = (testsssss * d_CONST_INT[e_d_CONST_INT_numRecordsPerThread] + numRecordsEnd)*e_d_IO_LAST;

  // d_IO[vi_IO_INT_OFFSET_END + e_d_IO_INT_RKQS_ERROR] = vi_TESTOTESTO;  
  // d_IO[vi_IO_INT_OFFSET_END + e_d_IO_INT_RKQS_STEPS] = l_rkqs_TRIED;
  
  d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_T] = l_time_CURRENT; 
  for (i = 0 ; i<4; i++)
  {
    d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_X + i] = l_spinor[i];
    d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_RED_VX + i] =   l_dspinordt[i];
    d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_SCAL_X + i] =   l_spinor_scal[i];
    d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_ERR_X + i] =   l_epsilon[i];
  }
  for (i = 0 ; i<3; i++)
  {
    d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_BX + i] = l_BField[i]; 
  }
  d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_POLARIZATION] = l_pol; 
  d_IO[vi_IO_DOUBLE_OFFSET_END + e_d_IO_HNEXT] = l_hnext; 
  
  return;
}
void GENERIC_RECORD_FRAME(
  int param_numBlocks,
  int param_numThreadsPerBlock,
  double *d_IO,
  int *d_IO_INT,
  int numRecordsStart,
  int numRecordsEnd)
{
  // cudaPrintfInit ();
    // Run kernel
  hipDeviceSynchronize();
  GENERIC_PIECEWISE_KERNEL_MULTI_XVS_RKQS_LOOP<<< param_numBlocks, param_numThreadsPerBlock >>>(
    d_IO, 
    d_IO_INT, 
    numRecordsStart,
    numRecordsEnd);
  // GENERIC_PIECEWISE_KERNEL_MULTI_XVS<<<h_CONST_INT[e_d_CONST_INT_numBlocks],h_CONST_INT[e_d_CONST_INT_numThreadsPerBlock]>>>(
    // d_IO, 
    // d_IO_INT, 
    // vi_RecordsStartCurrent,
    // vi_RecordsEndCurrent);
  // GENERIC_PIECEWISE_KERNEL_MULTI<<<h_CONST_INT[e_d_CONST_INT_numBlocks],h_CONST_INT[e_d_CONST_INT_numThreadsPerBlock]>>>(
    // d_IO, 
    // d_IO_INT, 
    // vi_RecordsStartCurrent,
    // vi_RecordsEndCurrent);
  // cudaPrintfDisplay(stdout,true);
  // cudaPrintEnd();
  hipDeviceSynchronize();
  return;
}

void GENERIC_MIDDLEMAN_MULTI(
  const double *h_CONST,
  const int *h_CONST_INT,
  double *h_IO,
  int *h_IO_INT)
{
  printf("\n\n\nelloel,lleoeooo%d shoudl be zero\n\n",e_RKQS_ERROR_NONE);
  // Establish Scope parameters for simulation: 
  // number of neutrons, for how long, how many records to keep, etc.
  
  // Copy passsed constant values from Host Memory (CPU front side bus RAM) to Device Constant Memory.
  // Device Constant Memory is limited in size but accessible with close to register level latency at the the thread level due to mandatory caching in every CUDA Multi-processor.
  
  int vi_INDEX;
  
  double h_UNOFFICIAL_CONST[e_d_CONST_LAST];
  for (vi_INDEX = 0; vi_INDEX<e_d_CONST_LAST; vi_INDEX++) h_UNOFFICIAL_CONST[vi_INDEX] = h_CONST[vi_INDEX];
  int h_UNOFFICIAL_CONST_INT[e_d_CONST_INT_LAST];
  for (vi_INDEX = 0; vi_INDEX<e_d_CONST_INT_LAST; vi_INDEX++) h_UNOFFICIAL_CONST_INT[vi_INDEX] = h_CONST_INT[vi_INDEX];
  
  const int numBytesCONST = e_d_CONST_LAST*sizeof(double);
  const int numBytesCONST_INT = e_d_CONST_INT_LAST*sizeof(int);
  
  int vi_ERROR = hipMemcpyToSymbol(HIP_SYMBOL(d_CONST),h_UNOFFICIAL_CONST,numBytesCONST);
  int vi_ERROR_INT = hipMemcpyToSymbol(HIP_SYMBOL(d_CONST_INT),h_UNOFFICIAL_CONST_INT,numBytesCONST_INT);
  
  // Move passed input parameters specific to each thread from Host Memory to Device Memory (on-card RAM)
  double *d_IO = NULL;
  int *d_IO_INT = NULL;
  const int numBytesIO = h_CONST_INT[e_d_CONST_INT_numRecords]*e_d_IO_LAST*sizeof(double);
  const int numBytesIO_INT = h_CONST_INT[e_d_CONST_INT_numRecords]*e_d_IO_INT_LAST*sizeof(int);
  hipMalloc((void**)&d_IO, numBytesIO);
  hipMalloc((void**)&d_IO_INT, numBytesIO_INT);
  hipMemcpy(d_IO, h_IO, numBytesIO, hipMemcpyHostToDevice);
  hipMemcpy(d_IO_INT, h_IO_INT, numBytesIO_INT, hipMemcpyHostToDevice);
  
  printf("\ncheck eeeee check check");
  hipDeviceSynchronize();
  // Allocate where output data goes with room for all threads
  // double *d_OUT = NULL;
  // int *d_OUT_INT = NULL;
  // const int numBytesOUT = h_CONST_INT[e_d_CONST_INT_numRecords]*e_d_IO_LAST*sizeof(double);
  // const int numBytesOUT_INT = h_CONST_INT[e_d_CONST_INT_numRecords]*e_d_IO_INT_LAST*sizeof(int);
  // hipMalloc((void**)&d_OUT, numBytesOUT);
  // hipMalloc((void**)&d_OUT_INT, numBytesOUT_INT);
  int vi_RecordsStartCurrent = 0;
  int vi_RecordsEndCurrent = 0;
  int vi_RecordsEndFinal = h_CONST_INT[e_d_CONST_INT_numRecordsPerThread];
  int vi_UPDATES;
  for (vi_UPDATES = 0; vi_RecordsEndCurrent<vi_RecordsEndFinal; vi_UPDATES++)
  {
    // Run kernel
    vi_RecordsStartCurrent = vi_RecordsEndCurrent;
    int vi_RecordsEndCurrent_TEMP =  vi_RecordsStartCurrent + h_CONST_INT[e_d_CONST_INT_numRecordsPerUpdate];
    if (vi_RecordsEndCurrent_TEMP>vi_RecordsEndFinal) vi_RecordsEndCurrent = vi_RecordsEndFinal;
    else vi_RecordsEndCurrent = vi_RecordsStartCurrent + h_CONST_INT[e_d_CONST_INT_numRecordsPerUpdate];
    // vi_RecordsStartCurrent = 0;
    // vi_RecordsEndCurrent = vi_RecordsEndFinal;
    printf("\nStarting Records %d-%d of %d... ",vi_RecordsStartCurrent,vi_RecordsEndCurrent,h_CONST_INT[e_d_CONST_INT_numRecordsPerThread]);
    GENERIC_RECORD_FRAME(
      h_CONST_INT[e_d_CONST_INT_numBlocks],
      h_CONST_INT[e_d_CONST_INT_numThreadsPerBlock],
      d_IO,
      d_IO_INT,
      vi_RecordsStartCurrent,
      vi_RecordsEndCurrent);
    // GENERIC_PIECEWISE_KERNEL_MULTI<<<h_CONST_INT[e_d_CONST_INT_numBlocks],h_CONST_INT[e_d_CONST_INT_numThreadsPerBlock]>>>(
      // d_IO, 
      // d_IO_INT, 
      // vi_RecordsStartCurrent,
      // vi_RecordsEndCurrent);
    printf("Completed");
  }
  printf("\nDay o day o daylight come and me wanna go %d threads",h_CONST_INT[e_d_CONST_INT_numThreads]);
  // Move results of output to Host Memory from Device Memory
  hipMemcpy(h_IO, d_IO, numBytesIO, hipMemcpyDeviceToHost);
  hipMemcpy(h_IO_INT, d_IO_INT, numBytesIO_INT, hipMemcpyDeviceToHost);
  
  hipFree(d_IO);
  hipFree(d_IO_INT);
  
  return;
}
